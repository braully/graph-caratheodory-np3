#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <string>
#include <sstream>
#include <dirent.h>
#include <unistd.h>
#include <fstream>
#include <sys/stat.h>
#include <sys/types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

#define CHARACTER_INIT_COMMENT '#'

#define DEFAULT_THREAD_PER_BLOCK 256
#define DEFAULT_BLOCK 256 
#define MAX_DEFAULT_SIZE_QUEUE 256
#define PROCESSED 3
#define INCLUDED 2
#define NEIGHBOOR_COUNT_INCLUDED 1
/* */
#define MAX(x, y) (((x) > (y)) ? (x) : (y))
#define MIN(x, y) (((x) < (y)) ? (x) : (y))

//extern __device__ int result[2];
__device__ int result_d;
__device__ volatile bool found = false;

__host__ __device__
int maxCombinations(int n, int k) {
    if (n == 0 || k == 0) {
        return 0;
    }
    if (n < k) {
        return 0;
    }
    if (n == k) {
        return 1;
    }
    int delta, idxMax;
    if (k < n - k) {
        delta = n - k;
        idxMax = k;
    } else {
        delta = k;
        idxMax = n - k;
    }

    int ans = delta + 1;
    for (int i = 2; i <= idxMax; ++i) {
        ans = (ans * (delta + i)) / i;
    }
    return ans;
}

__host__ __device__
void initialCombination(int n, int k, int* combinationArray, int idx) {
    int a = n;
    int b = k;
    int x = (maxCombinations(n, k) - 1) - idx;
    for (int i = 0; i < k; ++i) {
        combinationArray[i] = a - 1;
        while (maxCombinations(combinationArray[i], b) > x) {
            --combinationArray[i];
        }
        x = x - maxCombinations(combinationArray[i], b);
        a = combinationArray[i];
        b = b - 1;
    }

    for (int i = 0; i < k; ++i) {
        combinationArray[i] = (n - 1) - combinationArray[i];
    }
}

__host__ __device__
void initialCombination(int n, int k, int* combinationArray) {
    for (int i = 0; i < k; i++) {
        combinationArray[i] = i;
    }
}

__host__ __device__
void nextCombination(int n,
        int k,
        int* currentCombination) {
    if (currentCombination[0] == n - k) {
        return;
    }
    int i;
    for (i = k - 1; i > 0 && currentCombination[i] == n - k + i; --i);
    ++currentCombination[i];
    for (int j = i; j < k - 1; ++j) {
        currentCombination[j + 1] = currentCombination[j] + 1;
    }
}

__host__ __device__
void printCombination(int *currentCombination,
        int sizeComb) {
    printf("S = {");
    for (int i = 0; i < sizeComb; i++) {
        printf("%2d", currentCombination[i]);
        if (i < sizeComb - 1) {
            printf(", ");
        }
    }
    printf(" }");
}

__global__
void kernelFindCaratheodoryNumber(int nvertices, int *csrColIdxs, int sizeCsrColIdxs,
        int *csrRowOffset, int sizeRowOffset, int maxCombination, int k, int offset) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx == 0) {
        result_d = 0;
        found = false;
    }
    __syncthreads();

    //if (1)
    //    return;

    void *memspace = (void *) malloc(sizeof (int)*k + sizeof (int)*nvertices + sizeof (unsigned char)*nvertices);

    if (memspace == NULL) {
        printf("failed memory allocation\n");
        return;
    }

    int *currentCombinations = (int *) memspace;
    int* aux = (int *) (memspace + sizeof (int)*k);
    unsigned char *auxc = (unsigned char *) (memspace + sizeof (int)*k + sizeof (int)*nvertices);

    int sizederivated = 0;
    int limmit = (idx + 1) * offset;
    int k_i = idx * offset;

    if (limmit > maxCombination) {
        limmit = maxCombination;
    }

    initialCombination(nvertices, k, currentCombinations, k_i);

    while (k_i < limmit && !found) {
        //clean aux vector            
        for (int i = 0; i < nvertices; i++) {
            aux[i] = 0;
            auxc[i] = 0;
        }

        int headQueue = nvertices;
        int tailQueue = 0;

        for (int i = 0; i < k; i++) {
            int idi = currentCombinations[i];
            aux[idi] = INCLUDED;
            auxc[idi] = 1;
            headQueue = MIN(headQueue, idi);
            tailQueue = MAX(tailQueue, idi);
        }

        while (headQueue <= tailQueue) {
            int verti = headQueue;
            if (verti >= nvertices || aux[verti] != INCLUDED) {
                headQueue++;
                continue;
            }

            int end = csrColIdxs[verti + 1];
            for (int i = csrColIdxs[verti]; i < end; i++) {
                int vertn = csrRowOffset[i];
                if (vertn >= nvertices) continue;
                if (vertn != verti && aux[vertn] < INCLUDED) {
                    aux[vertn] = aux[vertn] + NEIGHBOOR_COUNT_INCLUDED;
                    if (aux[vertn] == INCLUDED) {
                        headQueue = MIN(headQueue, vertn);
                        tailQueue = MAX(tailQueue, vertn);
                    }
                    auxc[vertn] = auxc[vertn] + auxc[verti];
                }
            }
            aux[verti] = PROCESSED;
        }

        bool checkDerivated = false;

        for (int i = 0; i < nvertices; i++)
            if (auxc[i] >= k && aux[i] == PROCESSED) {
                checkDerivated = true;
                break;
            }

        if (checkDerivated) {
            for (int i = 0; i < k; i++) {
                int p = currentCombinations[i];
                headQueue = nvertices;
                tailQueue = -1;

                for (int j = 0; j < nvertices; j++) {
                    auxc[j] = 0;
                }

                for (int j = 0; j < k; j++) {
                    int v = currentCombinations[j];
                    if (v != p) {
                        auxc[v] = INCLUDED;
                        headQueue = MIN(headQueue, v);
                        tailQueue = MAX(tailQueue, v);
                    }
                }
                while (headQueue <= tailQueue) {
                    int verti = headQueue;

                    if (verti >= nvertices || auxc[verti] != INCLUDED) {
                        headQueue++;
                        continue;
                    }
                    aux[verti] = 0;
                    int end = csrColIdxs[verti + 1];
                    for (int x = csrColIdxs[verti]; x < end; x++) {
                        int vertn = csrRowOffset[x];
                        if (vertn != verti && auxc[vertn] < INCLUDED) {
                            auxc[vertn] = auxc[vertn] + NEIGHBOOR_COUNT_INCLUDED;
                            if (auxc[vertn] == INCLUDED) {
                                headQueue = MIN(headQueue, vertn);
                                tailQueue = MAX(tailQueue, vertn);
                            }
                        }
                    }
                    auxc[verti] = PROCESSED;
                }
            }
            sizederivated = 0;
            for (int i = 0; i < nvertices; i++)
                if (aux[i] >= INCLUDED) sizederivated++;
        }

        if (sizederivated == 0) {
            nextCombination(nvertices, k, currentCombinations);
            k_i++;
        } else {
            result_d = k_i + 1;
            found = true;
            printf("\nCartheodory Find - Thread-%d: sizederivated=%d k=%d k_i=%d",
                    idx, sizederivated, k, k_i);
        }
    }

    //    printf("\nThread-%d: szoffset=%d nvs=%d k=%d k_i=[%d, %d]=%d",
    //            idx, sizeRowOffset, nvertices, k, idx * offset, limmit, execCount);
    free(memspace);
}

bool checkIfCaratheodorySet() {
    for (int i = 0; i < nvertices; i++) {
        aux[i] = 0;
        auxc[i] = 0;
    }

    int headQueue = nvertices;
    int tailQueue = 0;

    for (int i = 0; i < k; i++) {
        int idi = currentCombinations[i];
        aux[idi] = INCLUDED;
        auxc[idi] = 1;
        headQueue = MIN(headQueue, idi);
        tailQueue = MAX(tailQueue, idi);
    }

    while (headQueue <= tailQueue) {
        int verti = headQueue;
        if (verti >= nvertices || aux[verti] != INCLUDED) {
            headQueue++;
            continue;
        }

        int end = csrColIdxs[verti + 1];
        for (int i = csrColIdxs[verti]; i < end; i++) {
            int vertn = csrRowOffset[i];
            if (vertn >= nvertices) continue;
            if (vertn != verti && aux[vertn] < INCLUDED) {
                aux[vertn] = aux[vertn] + NEIGHBOOR_COUNT_INCLUDED;
                if (aux[vertn] == INCLUDED) {
                    headQueue = MIN(headQueue, vertn);
                    tailQueue = MAX(tailQueue, vertn);
                }
                auxc[vertn] = auxc[vertn] + auxc[verti];
            }
        }
        aux[verti] = PROCESSED;
    }

    bool checkDerivated = false;
    bool isCaratheodory = false;

    for (int i = 0; i < nvertices; i++)
        if (auxc[i] >= k && aux[i] == PROCESSED) {
            checkDerivated = true;
            break;
        }

    if (checkDerivated) {
        for (int i = 0; i < k; i++) {
            int p = currentCombinations[i];
            headQueue = nvertices;
            tailQueue = -1;

            for (int j = 0; j < nvertices; j++) {
                auxc[j] = 0;
            }

            for (int j = 0; j < k; j++) {
                int v = currentCombinations[j];
                if (v != p) {
                    auxc[v] = INCLUDED;
                    headQueue = MIN(headQueue, v);
                    tailQueue = MAX(tailQueue, v);
                }
            }
            while (headQueue <= tailQueue) {
                int verti = headQueue;

                if (verti >= nvertices || auxc[verti] != INCLUDED) {
                    headQueue++;
                    continue;
                }
                aux[verti] = 0;
                int end = csrColIdxs[verti + 1];
                for (int x = csrColIdxs[verti]; x < end; x++) {
                    int vertn = csrRowOffset[x];
                    if (vertn != verti && auxc[vertn] < INCLUDED) {
                        auxc[vertn] = auxc[vertn] + NEIGHBOOR_COUNT_INCLUDED;
                        if (auxc[vertn] == INCLUDED) {
                            headQueue = MIN(headQueue, vertn);
                            tailQueue = MAX(tailQueue, vertn);
                        }
                    }
                }
                auxc[verti] = PROCESSED;
            }
        }
        sizederivated = 0;
        for (int i = 0; i < nvertices; i++)
            if (aux[i] >= INCLUDED) sizederivated++;
    }

    if (sizederivated == 0) {
        nextCombination(nvertices, k, currentCombinations);
        k_i++;
    } else {
        result_d = k_i + 1;
        found = true;
        printf("\nCartheodory Find - Thread-%d: sizederivated=%d k=%d k_i=%d",
                idx, sizederivated, k, k_i);
    }
    return isCaratheodory;
}

void findParallelCaratheodoryNumberBinaryStrategy(int verticesCount, int* csrColIdxs,
        int csrColIdxsSize, int* csrRowOffset, int sizeRowOffset) {
    int* csrColIdxsGpu;
    int* csrRowOffsetGpu;

    int numBytesClsIdx = sizeof (int)*csrColIdxsSize;
    hipMalloc((void**) &csrColIdxsGpu, numBytesClsIdx);

    int numBytesRowOff = sizeof (int)*sizeRowOffset;
    hipMalloc((void**) &csrRowOffsetGpu, numBytesRowOff);

    hipError_t r = hipMemcpy(csrColIdxsGpu, csrColIdxs, numBytesClsIdx, hipMemcpyHostToDevice);
    if (r != hipSuccess) {
        fprintf(stderr, "Failed to copy memory 1 \nError: %s\n", hipGetErrorString(r));
        exit(EXIT_FAILURE);
    }
    r = hipMemcpy(csrRowOffsetGpu, csrRowOffset, numBytesRowOff, hipMemcpyHostToDevice);
    if (r != hipSuccess) {
        fprintf(stderr, "Failed to copy memory 2 \nError: %s\n", hipGetErrorString(r));
        exit(EXIT_FAILURE);
    }

    int maxSizeSet = (verticesCount + 1) / 2;
    int k = 0;
    int left = 0;
    int rigth = maxSizeSet;
    int lastSize = -1;
    int lastIdxCaratheodorySet = -1;

    r = hipGetLastError();

    int result_h = 0;

    while (left <= rigth) {
        k = (left + rigth) / 2;
        int maxCombination = maxCombinations(verticesCount, k);
        int threadsPerBlock = MIN(ceil(maxCombination / 3.0), DEFAULT_THREAD_PER_BLOCK);
        int blocksPerGrid = MIN(ceil(maxCombination / (double) (threadsPerBlock) / 7), DEFAULT_BLOCK);
        int offset = ceil(maxCombination / (double) (blocksPerGrid * threadsPerBlock));
        //        http://stackoverflow.com/questions/34041372/access-cuda-global-device-variable-from-host

        if (k > 0) {
            printf("\nkernelFindCaratheodoryNumber: szoffset=%d nvs=%d k=%d max=%d blks=%d tdsPerBlock=%d offset=%d",
                    sizeRowOffset, verticesCount, k, maxCombination, blocksPerGrid, threadsPerBlock, offset);
            kernelFindCaratheodoryNumber << <blocksPerGrid, threadsPerBlock>>>(verticesCount, csrColIdxsGpu, csrColIdxsSize, csrRowOffsetGpu,
                    sizeRowOffset, maxCombination, k, offset);
        }

        r = hipDeviceSynchronize();
        if (r != hipSuccess) {
            fprintf(stderr, "Failed hipDeviceSynchronize \nError: %s\n", hipGetErrorString(r));
            exit(EXIT_FAILURE);
        }

        r = hipMemcpyFromSymbol(&result_h, HIP_SYMBOL(result_d), sizeof (int));
        if (r != hipSuccess) {
            fprintf(stderr, "Failed to copy memory 4 \nError: %s\n", hipGetErrorString(r));
            exit(EXIT_FAILURE);
        }

        r = hipGetLastError();
        if (r != hipSuccess) {
            fprintf(stderr, "Failed in kernelFindCaratheodoryNumber \nError: %s\n", hipGetErrorString(r));
            exit(EXIT_FAILURE);
        }

        if (result_h > 0) {
            lastSize = k;
            lastIdxCaratheodorySet = result_h - 1;
            left = k + 1;
            result_h = 0;
            printf("\nCaratheodory set size=%d..Ok idx=%d", k, lastIdxCaratheodorySet);
        } else {
            rigth = k - 1;
            printf("\nCaratheodory set size=%d..Not", k);
        }
    }

    if (lastSize > 0) {
        printf("\nResult Parallel Binary\n");
        int *currentCombination = (int *) malloc(lastSize * sizeof (int));
        initialCombination(verticesCount, lastSize, currentCombination, lastIdxCaratheodorySet);
        printCombination(currentCombination, lastSize);
        printf("\nS=%d-Comb(%d,%d) \n|S| = %d\n",
                lastIdxCaratheodorySet, verticesCount, lastSize, lastSize);
    } else {
        printf("\nCaratheodory set not found!");
    }
    hipFree(csrRowOffsetGpu);
    hipFree(csrColIdxsGpu);
}

int main(int argc, char** argv) {
    char* strFile = "graph-test/graph-csr-8775879977288244551.txt";
    bool parallel = false;
    bool binary = false;

    if ((argc <= 1) || (argv[argc - 1] == NULL) || (argv[argc - 1][0] == '-')) {
        parallel = true;
        binary = true;
    } else {
        strFile = argv[1];
    }

    DIR *dpdf;
    struct dirent *epdf;
    struct stat filestat;

    dpdf = opendir(strFile);
    std::string filepath = std::string(strFile);
    while (dpdf && (epdf = readdir(dpdf))) {
        filepath = std::string(strFile) + "/" + epdf->d_name;
        if (epdf->d_name == "." || epdf->d_name == "..")
            continue;
        if (stat(filepath.c_str(), &filestat))
            continue;
        if (S_ISDIR(filestat.st_mode))
            continue;
        else break;
    }
    closedir(dpdf);

    std::string line, strCArray, strRArray;
    std::ifstream infile(filepath.c_str());

    if (infile) {
        while (getline(infile, line)) {
            if (line.at(0) != CHARACTER_INIT_COMMENT) {
                if (strCArray.empty()) {
                    strCArray = line;
                } else if (strRArray.empty()) {
                    strRArray = line;
                }
            }
        }
        infile.close();
    } else {
        printf("file '%s' not found!", filepath.c_str());
        return;
    }

    if (strCArray.empty() || strRArray.empty()) {
        perror("Invalid file format");
        return;
    }

    std::stringstream stream(strCArray.c_str());
    std::vector<int> values;

    int n;
    while (stream >> n) {
        values.push_back(n);
    }
    strCArray.clear();

    int numVertices = values.size() - 1;
    int *colIdx = new int[numVertices + 1];
    std::copy(values.begin(), values.end(), colIdx);
    values.clear();
    stream.str("");

    std::stringstream stream2(strRArray);
    while (stream2 >> n) {
        values.push_back(n);
    }
    stream2.str("");
    strRArray.clear();

    int sizeRowOffset = values.size();
    int *rowOffset = new int[sizeRowOffset];
    std::copy(values.begin(), values.end(), rowOffset);
    values.clear();

    printf("\nProcess file: %s", filepath.c_str());
    findParallelCaratheodoryNumberBinaryStrategy(numVertices, colIdx, sizeRowOffset, rowOffset, sizeRowOffset);
    return 0;
}

