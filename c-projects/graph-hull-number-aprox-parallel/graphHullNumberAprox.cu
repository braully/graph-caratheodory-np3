#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <string>
#include <sstream>
#include <dirent.h>
#include <unistd.h>
#include <fstream>
#include <sys/stat.h>
#include <sys/types.h>
#include <vector>
#include <math.h> 
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "nvgraph.h"

#define CHARACTER_INIT_COMMENT '#'

#define DEFAULT_THREAD_PER_BLOCK 256
#define DEFAULT_BLOCK 256 
#define MAX_DEFAULT_SIZE_QUEUE 256

#define SINCLUDED 4
#define PROCESSED 3
#define INCLUDED 2
#define NEIGHBOOR_COUNT_INCLUDED 1
/* */
#define MAX(x, y) (((x) > (y)) ? (x) : (y))
#define MIN(x, y) (((x) < (y)) ? (x) : (y))
#define COPY_ARRAY(SRC,DST,LEN) { memcpy((DST), (SRC), LEN); }

#define verboseSerial false

typedef struct nvgraphCSRTopology32I_st graphCsr;

__host__ __device__
int addVertToS(int vert, unsigned char* aux, graphCsr *graph) {
    int countIncluded = 0;
    int nvertices = graph->nvertices;

    if (aux[vert] >= INCLUDED) {
        return countIncluded;
    }
    int headQueue = vert;
    int tailQueue = vert;
    aux[vert] = INCLUDED;

    while (headQueue <= tailQueue) {
        int verti = headQueue;
        if (verti >= nvertices || aux[verti] != INCLUDED) {
            headQueue++;
            continue;
        }
        int end = graph->destination_indices[verti + 1];
        for (int j = graph->destination_indices[verti]; j < end; j++) {
            int vertn = graph->source_offsets[j];
            if (vertn >= nvertices) continue;
            if (vertn != verti && aux[vertn] < INCLUDED) {
                aux[vertn] = aux[vertn] + NEIGHBOOR_COUNT_INCLUDED;
                if (aux[vertn] == INCLUDED) {
                    headQueue = MIN(headQueue, vertn);
                    tailQueue = MAX(tailQueue, vertn);
                }
            }
        }
        aux[verti] = PROCESSED;
        countIncluded++;
    }
    aux[vert] = SINCLUDED;
    return countIncluded;
}

__global__
void kernelSerialAproxHullNumber(graphCsr *graphs, int* results) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    graphCsr *graph = &graphs[idx];
    int nvertices = graph->nvertices;
    unsigned char *aux = new unsigned char [nvertices];
    unsigned char *auxb = new unsigned char [nvertices];
    int minHullSet = nvertices;

    for (int v = 0; v < nvertices; v++) {
        for (int j = 0; j < nvertices; j++) {
            aux[j] = 0;
        }

        int sizeHs = addVertToS(v, aux, graph);
        int sSize = 1;
        int bv;
        do {
            bv = -1;
            int maiorGrau = 0;
            int maiorDeltaHs = 0;
            for (int i = 0; i < nvertices; i++) {
                if (aux[i] >= INCLUDED) {
                    continue;
                }
                COPY_ARRAY(aux, auxb, nvertices);
                int deltaHsi = addVertToS(i, auxb, graph);

                int neighborCount = 0;
                for (int j = 0; j < nvertices; j++) {
                    if (auxb[j] == INCLUDED) {
                        neighborCount++;
                    }
                }

                if (bv == -1 || (deltaHsi >= maiorDeltaHs && neighborCount > maiorGrau)) {
                    maiorDeltaHs = deltaHsi;
                    maiorGrau = neighborCount;
                    bv = i;
                }
            }
            sizeHs = sizeHs + addVertToS(bv, aux, graph);
            sSize++;
        } while (sizeHs < nvertices);
        minHullSet = MIN(minHullSet, sSize);
    }
    free(aux);
    free(auxb);
    results[idx] = minHullSet;
}

int serialAproxHullNumberGraphs(graphCsr *graphs, int cont) {
    printf("serialAproxHullNumberGraphs\n");
    int numbytesDataGraph = 0;
    hipError_t r;
    for (int i = 0; i < cont; i++) {
        int size = (graphs[i].nvertices + 1) * sizeof (int);
        size = size + graphs[i].nedges * sizeof (int);
        numbytesDataGraph = numbytesDataGraph + size;
    }

    printf("Cuda Malloc Graph\n");

    int* dataGraphsGpu;
    hipMalloc((void**) &dataGraphsGpu, numbytesDataGraph);

    int* resultGpu;
    hipMalloc((void**) &resultGpu, cont * sizeof (int));

    graphCsr *graphsGpu;
    hipMalloc((void**) &graphsGpu, cont * sizeof (graphCsr));
    r = hipMemcpy(graphsGpu, graphs, cont * sizeof (graphCsr), hipMemcpyHostToDevice);

    int offset = 0;

    printf("Cuda Atrib Graph\n");
    for (int i = 0; i < cont; i++) {
        graphCsr *graph = &graphs[i];
        graphCsr *graphGpu = &graphsGpu[i];
        int nbytes = (graph->nvertices + 1) * sizeof (int);
        r = hipMemcpy(dataGraphsGpu + offset, graph->destination_indices, nbytes, hipMemcpyHostToDevice);
        if (r != hipSuccess) {
            fprintf(stderr, "Failed to copy memory 1 \nError: %s\n", hipGetErrorString(r));
            exit(EXIT_FAILURE);
        }
        graphGpu->destination_indices = dataGraphsGpu + offset;

        offset = offset + nbytes;

        nbytes = graph->nedges * sizeof (int);

        r = hipMemcpy(dataGraphsGpu + offset, graph->source_offsets, nbytes, hipMemcpyHostToDevice);
        if (r != hipSuccess) {
            fprintf(stderr, "Failed to copy memory 1 \nError: %s\n", hipGetErrorString(r));
            exit(EXIT_FAILURE);
        }
        graphGpu->source_offsets = dataGraphsGpu + offset;
        offset = offset + graph->nedges;
    }


    printf("Launch Kernel\n");

    //    kernelSerialAproxHullNumber << <1, cont>>>(graphsGpu, resultGpu);

    printf("Read Result\n");

    //    r = hipDeviceSynchronize();
    if (r != hipSuccess) {
        fprintf(stderr, "Failed hipDeviceSynchronize \nError: %s\n", hipGetErrorString(r));
        exit(EXIT_FAILURE);
    }


    //    int* resultLocal;

    //    r = hipMemcpyFromSymbol(&resultLocal, HIP_SYMBOL(resultGpu), sizeof (int)*cont);
    if (r != hipSuccess) {
        fprintf(stderr, "Failed to copy memory 4 \nError: %s\n", hipGetErrorString(r));
        exit(EXIT_FAILURE);
    }

    r = hipGetLastError();
    if (r != hipSuccess) {
        fprintf(stderr, "Failed in kernelSerialAproxHullNumber \nError: %s\n", hipGetErrorString(r));
        exit(EXIT_FAILURE);
    }

    for (int i = 0; i < cont; i++) {
        //        printf("MinHullNumberAprox Graph-%d: %d\n", i, resultLocal[i]);
    }

}

void processFiles(int argc, char** argv) {
    graphCsr* graphs = (graphCsr*) malloc((sizeof (graphCsr)) * argc);
    std::vector<int> values;

    int contGraph = 0;

    for (int x = 1; x < argc; x++) {
        char* strFile = "graph-test.txt";
        strFile = argv[x];
        DIR *dpdf;
        struct dirent *epdf;
        struct stat filestat;

        dpdf = opendir(strFile);
        std::string filepath = std::string(strFile);

        while (dpdf && (epdf = readdir(dpdf))) {
            filepath = std::string(strFile) + "/" + epdf->d_name;
            if (epdf->d_name == "." || epdf->d_name == "..")
                continue;
            if (stat(filepath.c_str(), &filestat))
                continue;
            if (S_ISDIR(filestat.st_mode))
                continue;
            else break;
        }
        closedir(dpdf);

        std::string line, strCArray, strRArray;
        std::ifstream infile(filepath.c_str());

        if (infile) {
            while (getline(infile, line)) {
                if (line.at(0) != CHARACTER_INIT_COMMENT) {
                    if (strCArray.empty()) {
                        strCArray = line;
                    } else if (strRArray.empty()) {
                        strRArray = line;
                    }
                }
            }
            infile.close();
        } else {
            continue;
        }

        if (strCArray.empty() || strRArray.empty()) {
            perror("Invalid file format");
            continue;
        }

        std::stringstream stream(strCArray.c_str());
        values.clear();
        int n;
        while (stream >> n) {
            values.push_back(n);
        }
        strCArray.clear();

        int numVertices = values.size() - 1;
        int *colIdx = new int[numVertices + 1];
        std::copy(values.begin(), values.end(), colIdx);
        values.clear();
        stream.str("");

        std::stringstream stream2(strRArray);
        while (stream2 >> n) {

            values.push_back(n);
        }
        stream2.str("");
        strRArray.clear();

        int sizeRowOffset = values.size();
        int *rowOffset = new int[sizeRowOffset];
        std::copy(values.begin(), values.end(), rowOffset);
        values.clear();

        //        graphCsr* graph = (graphCsr*) malloc(sizeof (graphCsr));
        graphCsr* graph = &graphs[contGraph];
        graph->nvertices = numVertices;
        graph->destination_indices = colIdx;
        graph->source_offsets = rowOffset;
        contGraph++;
    }
    serialAproxHullNumberGraphs(graphs, contGraph);

}

void runTest() {

    int numVertices = 10;
    int colIdx[] = {0, 3, 6, 9, 12, 15, 18, 21, 24, 27, 30};
    int sizeRowOffset = numVertices + 1;
    int rowOffset[] = {2, 5, 6, 3, 4, 6, 0, 4, 7, 1, 5, 7, 1, 2, 9, 0, 3, 9, 0, 1, 8, 2, 3, 8, 6, 7, 9, 4, 5, 8};
    graphCsr* graph = (graphCsr*) malloc(sizeof (graphCsr));
    graph->nvertices = numVertices;
    graph->nedges = 30;
    graph->destination_indices = colIdx;
    graph->source_offsets = rowOffset;
    //    int minSerialAprox = serialAproxHullNumber(graph);
    //    printf("MinAproxHullSet: %d\n", minSerialAprox);

    serialAproxHullNumberGraphs(graph, 1);
}

int main(int argc, char** argv) {
    printf("Main\n");
    if (argc > 1) {
        processFiles(argc, argv);
    } else {
        runTest();
    }
    return 0;
}