#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <string>
#include <sstream>
#include <dirent.h>
#include <unistd.h>
#include <fstream>
#include <sys/stat.h>
#include <sys/types.h>
#include <vector>
#include <math.h> 
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <limits.h>
#include "nvgraph.h"

#define CHARACTER_INIT_COMMENT '#'

#define BLOCK_WINDOWS 32
#define BLOCK_SIZE_OPTIMAL 256
#define BLOCK_FACTOR_OPTIMAL 2

#define DEFAULT_THREAD_PER_BLOCK 28
#define DEFAULT_BLOCK 256 
#define MAX_DEFAULT_SIZE_QUEUE 256

#define WARP_SIZE 32
#define SINCLUDED 4
#define PROCESSED 3
#define INCLUDED 2
#define NEIGHBOOR_COUNT_INCLUDED 1
/* */
#define MAX(x, y) (((x) > (y)) ? (x) : (y))
#define MIN(x, y) (((x) < (y)) ? (x) : (y))
#define COPY_ARRAY(SRC,DST,LEN) { memcpy((DST), (SRC), LEN); }

#define verboseKernel false

bool verbose = false;
bool graphByThread = false;
bool graphByBlock = false;
bool graphByBlockOptimal = false;
bool verticesBythread = false;
bool serial = false;
bool printResults = true;

struct graphCsr {
    int *data;
    //    data[0] nVertices;
    //    data[0] nEdges
    //    data[2] *csrColIdxs;
    //    data[nVertices+3] *csrRowOffset;
};

__host__ __device__
int addVertToS(int vert, unsigned char* aux, int *graphData) {
    int countIncluded = 0;
    int nvertices = graphData[0];
    int *csrColIdxs = &graphData[2];
    int *csrRowOffset = &graphData[nvertices + 3];

    if (aux[vert] >= INCLUDED) {
        return countIncluded;
    }
    int headQueue = vert;
    int tailQueue = vert;
    aux[vert] = INCLUDED;

    while (headQueue <= tailQueue) {
        int verti = headQueue;
        if (verti >= nvertices || aux[verti] != INCLUDED) {
            headQueue++;
            continue;
        }
        int end = csrColIdxs[verti + 1];
        for (int j = csrColIdxs[verti]; j < end; j++) {
            int vertn = csrRowOffset[j];
            if (vertn >= nvertices) continue;
            if (vertn != verti && aux[vertn] < INCLUDED) {
                aux[vertn] = aux[vertn] + NEIGHBOOR_COUNT_INCLUDED;
                if (aux[vertn] == INCLUDED) {
                    headQueue = MIN(headQueue, vertn);
                    tailQueue = MAX(tailQueue, vertn);
                }
            }
        }
        aux[verti] = PROCESSED;
        countIncluded++;
    }
    aux[vert] = SINCLUDED;
    return countIncluded;
}

__host__ __device__
int exapandHullSetFromV(int v, int nvertices, unsigned char *aux, unsigned char *auxb, int *graphData, int idx) {
    for (int j = 0; j < nvertices; j++) {
        aux[j] = 0;
    }
    if (verboseKernel) printf("thread-%d: cleanded aux\n", idx);
    if (verboseKernel) printf("thread-%d: add Vert %d to S\n", idx, v);

    int sizeHs = addVertToS(v, aux, graphData);
    int sSize = 1;
    int bv;
    do {
        bv = -1;
        int maiorGrau = 0;
        int maiorDeltaHs = 0;
        for (int i = 0; i < nvertices; i++) {
            if (aux[i] >= INCLUDED) {
                continue;
            }
            COPY_ARRAY(aux, auxb, nvertices);
            int deltaHsi = addVertToS(i, auxb, graphData);

            int neighborCount = 0;
            for (int j = 0; j < nvertices; j++) {
                if (auxb[j] == INCLUDED) {
                    neighborCount++;
                }
            }

            if (bv == -1 || (deltaHsi >= maiorDeltaHs && neighborCount > maiorGrau)) {
                maiorDeltaHs = deltaHsi;
                maiorGrau = neighborCount;
                bv = i;
            }
        }
        if (verboseKernel) printf("thread-%d: add Vert %d to S\n", idx, bv);
        sizeHs = sizeHs + addVertToS(bv, aux, graphData);
        sSize++;
    } while (sizeHs < nvertices);
    return sSize;
}

__global__
void kernelAproxHullNumberGraphByBlock(int* graphsGpu, int* dataGraphs, int* results) {
    //void kernelAproxHullNumber(graphCsr *graphs, int* results) {
    int offset = blockIdx.x;
    int idx = threadIdx.x;
    if (verboseKernel) printf("thread-%d\n", idx);

    int start = graphsGpu[offset];
    if (verboseKernel) printf("thread-%d Graph-start: %d\n", idx, start);

    int *graphData = &dataGraphs[start];
    int nvertices = graphData[0];

    if (idx == 0) {
        results[offset] = nvertices;
    }
    __syncthreads();

    if (verboseKernel) printf("thread-%d Graph-nvertices: %d\n", idx, nvertices);
    if (verboseKernel) printf("thread-%d Graph-edges: %d\n", idx, graphData[1]);


    if (idx < nvertices) {
        unsigned char *aux = new unsigned char [nvertices * 2];
        unsigned char *auxb = &aux[nvertices];

        int v = idx;
        int minHullSet = nvertices;
        int sSize = exapandHullSetFromV(v, nvertices, aux, auxb, graphData, idx);
        minHullSet = MIN(minHullSet, sSize);
        atomicMin(&results[offset], minHullSet);

        if (verboseKernel) printf("thread-%d: minHullSet %d\n", idx, minHullSet);

        free(aux);
        //    free(auxb);
        if (verboseKernel) printf("thread-%d: memory freed\n", idx);
    }
}

__global__
void kernelAproxHullNumberByVertex(int offset, int* graphsGpu,
        int* dataGraphs, int* results) {
    //void kernelAproxHullNumber(graphCsr *graphs, int* results) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (verboseKernel)
        printf("thread-%d\n", idx);

    int start = graphsGpu[offset];
    if (verboseKernel)
        printf("thread-%d Graph-start: %d\n", idx, start);

    int *graphData = &dataGraphs[start];
    int nvertices = graphData[0];

    if (idx == 0) {
        results[offset] = nvertices;
    }
    __syncthreads();

    if (idx < nvertices) {
        if (verboseKernel)
            printf("thread-%d Graph-nvertices: %d\n", idx, nvertices);
        if (verboseKernel)
            printf("thread-%d Graph-edges: %d\n", idx, graphData[1]);

        unsigned char *aux = new unsigned char [nvertices * 2];
        unsigned char *auxb = &aux[nvertices];
        int minHullSet = nvertices;
        int v = idx;

        int sSize = exapandHullSetFromV(v, nvertices, aux, auxb, graphData, idx);
        minHullSet = MIN(minHullSet, sSize);

        if (verboseKernel)
            printf("thread-%d: minHullSet %d\n", idx, minHullSet);
        if (verboseKernel)
            printf("thread-%d: memory freed\n", idx);
        atomicMin(&results[offset], minHullSet);
        free(aux);
    }
}

__global__
void kernelAproxHullNumberGraphByThread(int* graphsGpu, int* dataGraphs, int* results) {
    //void kernelAproxHullNumber(graphCsr *graphs, int* results) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (verboseKernel)
        printf("thread-%d\n", idx);

    int start = graphsGpu[idx];
    if (verboseKernel)
        printf("thread-%d Graph-start: %d\n", idx, start);

    int *graphData = &dataGraphs[start];
    int nvertices = graphData[0];

    if (verboseKernel)
        printf("thread-%d Graph-nvertices: %d\n", idx, nvertices);
    if (verboseKernel)
        printf("thread-%d Graph-edges: %d\n", idx, graphData[1]);

    unsigned char *aux = new unsigned char [nvertices * 2];
    //    unsigned char *auxb = new unsigned char [nvertices];
    unsigned char *auxb = &aux[nvertices];

    int minHullSet = nvertices;

    for (int v = 0; v < nvertices; v++) {
        for (int j = 0; j < nvertices; j++) {
            aux[j] = 0;
        }
        if (verboseKernel) printf("thread-%d: cleanded aux\n", idx);
        if (verboseKernel) printf("thread-%d: add Vert %d to S\n", idx, v);

        int sizeHs = addVertToS(v, aux, graphData);
        int sSize = 1;
        int bv;
        do {
            bv = -1;
            int maiorGrau = 0;
            int maiorDeltaHs = 0;
            for (int i = 0; i < nvertices; i++) {
                if (aux[i] >= INCLUDED) {
                    continue;
                }
                COPY_ARRAY(aux, auxb, nvertices);
                int deltaHsi = addVertToS(i, auxb, graphData);

                int neighborCount = 0;
                for (int j = 0; j < nvertices; j++) {
                    if (auxb[j] == INCLUDED) {
                        neighborCount++;
                    }
                }

                if (bv == -1 || (deltaHsi >= maiorDeltaHs && neighborCount > maiorGrau)) {
                    maiorDeltaHs = deltaHsi;
                    maiorGrau = neighborCount;
                    bv = i;
                }
            }
            if (verboseKernel) printf("thread-%d: add Vert %d to S\n", idx, bv);
            sizeHs = sizeHs + addVertToS(bv, aux, graphData);
            sSize++;
        } while (sizeHs < nvertices && sSize < minHullSet);
        minHullSet = MIN(minHullSet, sSize);
        if (verboseKernel) printf("thread-%d: minHullSet %d\n", idx, minHullSet);
    }
    free(aux);
    //    free(auxb);
    if (verboseKernel) printf("thread-%d: memory freed\n", idx);
    results[idx] = minHullSet;
}


//kernelAproxHullNumberGraphByBlockOptimal<<<numblocks,nthreads,maxgraphsbyblock*sizeof(int)>>>(graphsGpu, dataGraphsGpu, resultGpu, minvertice, maxvertice, totalvertices, maxgraphsbyblock);

__global__
void kernelAproxHullNumberGraphByBlockOptimal(int *idxGraphsGpu, int* graphsGpu, int ngraphs, int* dataGraphs, int* results, int minvertice, int maxvertice, int totalvertices, int maxgraphsbyblock) {
    //void kernelAproxHullNumber(graphCsr *graphs, int* results) {
    int offset = blockIdx.x;
    int idx = threadIdx.x;
    __shared__ int g[64];

    if (idx == 0) {
        int cont = blockIdx.x;
        while (idxGraphsGpu[cont] < blockIdx.x) {
            cont++;
        }
        offset = cont;
        int *graphData = &dataGraphs[offset];
        int nvertices = graphData[0];
        int proxcont = cont;
        int i = 0;
        g[i] = idxGraphsGpu[proxcont];
        results[proxcont] = nvertices;
        proxcont++;

        while (idxGraphsGpu[cont] == idxGraphsGpu[proxcont]) {
            graphData = &dataGraphs[proxcont];
            nvertices = graphData[0];
            results[proxcont] = nvertices;
            g[i] = idxGraphsGpu[proxcont];
            proxcont++;
        }
    }
    __syncthreads();
    
    


    //Search position graph
    //    int idxvert = threadIdx.x;
    //    int f = 0;
    //    int l = ngraphs - 1;
    //    int m = (f + l) / 2;
    //
    //    while (f <= l) {
    //        if (graphsGpu[m] < search)
    //            first = middle + 1;
    //        else if (array[middle] == search) {
    //            printf("%d found at location %d.\n", search, middle + 1);
    //            break;
    //        } else
    //            last = middle - 1;
    //
    //        middle = (first + last) / 2;
    //    }

    if (verboseKernel) printf("thread-%d\n", idx);

    int start = graphsGpu[offset];
    if (verboseKernel) printf("thread-%d Graph-start: %d\n", idx, start);


    int *graphData = &dataGraphs[start];
    int nvertices = graphData[0];


    if (verboseKernel) printf("thread-%d Graph-nvertices: %d\n", idx, nvertices);
    if (verboseKernel) printf("thread-%d Graph-edges: %d\n", idx, graphData[1]);


    if (idx < nvertices) {
        unsigned char *aux = new unsigned char [nvertices * 2];
        unsigned char *auxb = &aux[nvertices];

        int v = idx;
        int minHullSet = nvertices;
        int sSize = exapandHullSetFromV(v, nvertices, aux, auxb, graphData, idx);
        minHullSet = MIN(minHullSet, sSize);
        atomicMin(&results[offset], minHullSet);

        if (verboseKernel) printf("thread-%d: minHullSet %d\n", idx, minHullSet);

        free(aux);
        //    free(auxb);
        if (verboseKernel) printf("thread-%d: memory freed\n", idx);
    }
}

int parallelAproxHullNumberGraphs(graphCsr *graphs, int cont) {
    if (verbose) printf("ParallelAproxHullNumberGraphs\n");
    int numbytesDataGraph = 0;
    hipError_t r;
    for (int i = 0; i < cont; i++) {
        int size = (graphs[i].data[0] + 3 + graphs[i].data[1]) * sizeof (int);
        numbytesDataGraph = numbytesDataGraph + size;
    }

    if (verbose) printf("Cuda Malloc Graph\n");

    int device;
    hipDeviceProp_t deviceProp;
    hipGetDevice(&device);
    hipGetDeviceProperties(&deviceProp, device);

    if ((deviceProp.concurrentKernels == 0)) {
        perror("not support for concurrent kernel\n");
    }
    if (verbose)
        printf("Compute SM %d.%d with %d multi-processors\n", deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount);

    int* dataGraphsGpu;
    int *graphsGpu;
    int *idxGraphsGpu;
    int* resultGpu;
    int* graphsHost = new int[cont];
    int* idxGraphsHost = new int[cont];
    hipMalloc((void**) &resultGpu, cont * sizeof (int));
    hipMalloc((void**) &dataGraphsGpu, numbytesDataGraph);
    hipMalloc((void**) &graphsGpu, cont * sizeof (int));
    hipMalloc((void**) &idxGraphsGpu, cont * sizeof (int));
    r = hipMemcpy(graphsGpu, graphs, cont * sizeof (int), hipMemcpyHostToDevice);
    int offset = 0;
    int totalVert = 0;
    int currentCont = 0;

    //if (verbose) printf("Cuda Atrib Graph\n");
    for (int i = 0; i < cont; i++) {
        //if (verbose) printf("Cuda Atrib Graph-%d\n", i);
        graphCsr *graph = &graphs[i];
        int sizeGraph = graph->data[0] + 3 + graph->data[1];
        int nbytes = sizeGraph * sizeof (int);
        r = hipMemcpy(dataGraphsGpu + offset, graph->data, nbytes, hipMemcpyHostToDevice);
        if (r != hipSuccess) {
            fprintf(stderr, "Failed to copy memory 1 \nError: %s\n", hipGetErrorString(r));
            exit(EXIT_FAILURE);
        }
        graphsHost[i] = offset;
        offset = offset + sizeGraph;

        idxGraphsHost[i] = currentCont;
        currentCont = currentCont + (totalVert / BLOCK_WINDOWS);
        totalVert = (totalVert + sizeGraph) % BLOCK_WINDOWS;
    }

    r = hipMemcpy(graphsGpu, idxGraphsHost, cont * sizeof (int), hipMemcpyHostToDevice);
    if (r != hipSuccess) {
        fprintf(stderr, "Failed to copy memory 2 \nError: %s\n", hipGetErrorString(r));
        exit(EXIT_FAILURE);
    }

    r = hipMemcpy(idxGraphsGpu, graphsHost, cont * sizeof (int), hipMemcpyHostToDevice);
    if (r != hipSuccess) {
        fprintf(stderr, "Failed to copy memory 1.2 \nError: %s\n", hipGetErrorString(r));
        exit(EXIT_FAILURE);
    }

    hipStream_t *streams = (hipStream_t *) malloc(cont * sizeof (hipStream_t));
    for (int i = 0; i < cont; i++) {
        hipStreamCreate(&(streams[i]));
    }

    if (verbose) printf("Launch Kernel\n");

    int* resultLocal = new int[cont];

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    /* Graphs by Thread */
    if (graphByThread) {
        if (verbose) printf("Lanch Kernel One graph by thread\n");

        hipEventRecord(start);

        kernelAproxHullNumberGraphByThread << <1, cont>>>(graphsGpu, dataGraphsGpu, resultGpu);
        r = hipDeviceSynchronize();

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("Time for the kernel(one graph by thread): %2.f ms\n", milliseconds);

        if (verbose) printf("Read Result\n");

        if (r != hipSuccess) {
            fprintf(stderr, "Failed hipDeviceSynchronize \nError: %s\n", hipGetErrorString(r));
            exit(EXIT_FAILURE);
        }

        r = hipMemcpy(resultLocal, resultGpu, sizeof (int)*cont, hipMemcpyDeviceToHost);
        if (r != hipSuccess) {
            fprintf(stderr, "Failed to copy memory 4 \nError: %s\n", hipGetErrorString(r));
            exit(EXIT_FAILURE);
        }

        if (printResults)
            for (int i = 0; i < cont; i++) {
                printf("MinHullNumberAprox-Thread Graph-%d: %d\n", i, resultLocal[i]);
            }
    }

    /* Graphs by Block */
    if (graphByBlock) {
        if (verbose)
            printf("Lauch Kernel One graph by Block\n");
        hipEventRecord(start);

        int maxvertice = 0;
        for (int i = 0; i < cont; i++) {
            graphCsr *graph = &graphs[i];
            int nvertice = graph->data[0];
            if (nvertice > maxvertice) {
                maxvertice = nvertice;
            }
        }

        kernelAproxHullNumberGraphByBlock << <cont, maxvertice>>>(graphsGpu, dataGraphsGpu, resultGpu);

        //        r = hipDeviceSynchronize();

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("Time for the (one graph by block): %2.f ms\n", milliseconds);

        if (verbose)
            printf("Read Result\n");


        if (r != hipSuccess) {
            fprintf(stderr, "Failed hipDeviceSynchronize \nError: %s\n", hipGetErrorString(r));
            exit(EXIT_FAILURE);
        }


        r = hipMemcpy(resultLocal, resultGpu, sizeof (int)*cont, hipMemcpyDeviceToHost);
        if (r != hipSuccess) {
            fprintf(stderr, "Failed to copy memory 4 \nError: %s\n", hipGetErrorString(r));
            exit(EXIT_FAILURE);
        }

        if (printResults)
            for (int i = 0; i < cont; i++) {
                printf("MinHullNumberAprox-Block Graph-%d: %d\n", i, resultLocal[i]);
            }
    }

    /* Graph by Grid and Vertices by Thread */
    if (verticesBythread) {
        if (verbose)
            printf("Vertices by Thread One graph by kernel\n");
        hipEventRecord(start);

        for (int i = 0; i < cont; i++) {
            graphCsr *graph = &graphs[i];
            int nvertice = graph->data[0];
            int nblocks = nvertice / DEFAULT_THREAD_PER_BLOCK;
            if ((nvertice % DEFAULT_THREAD_PER_BLOCK) > 0) {
                nblocks++;
            }
            //            int nthreads = MIN(DEFAULT_THREAD_PER_BLOCK, nvertice);
            int nthreads = DEFAULT_THREAD_PER_BLOCK;
            kernelAproxHullNumberByVertex << <nblocks, nthreads, 0, streams[i]>>>(i, graphsGpu, dataGraphsGpu, resultGpu);
        }
        //        r = hipDeviceSynchronize();

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("Time for Vt/Thread kernel(one graph by kernel): %2.f ms\n", milliseconds);

        if (verbose)
            printf("Read Result\n");


        if (r != hipSuccess) {
            fprintf(stderr, "Failed hipDeviceSynchronize \nError: %s\n", hipGetErrorString(r));
            exit(EXIT_FAILURE);
        }


        r = hipMemcpy(resultLocal, resultGpu, sizeof (int)*cont, hipMemcpyDeviceToHost);
        if (r != hipSuccess) {
            fprintf(stderr, "Failed to copy memory 4 \nError: %s\n", hipGetErrorString(r));
            exit(EXIT_FAILURE);
        }

        if (printResults)
            for (int i = 0; i < cont; i++) {
                printf("MinHullNumberAprox-Vertices Thread-%d: %d\n", i, resultLocal[i]);
            }
    }

    /* Graphs by Block Optimal */
    if (graphByBlockOptimal) {
        if (verbose)
            printf("Lauch Kernel graph by Block optimal\n");
        hipEventRecord(start);

        int maxvertice = 0;
        int minvertice = INT_MAX;
        int totalvertices = 0;

        for (int i = 0; i < cont; i++) {
            graphCsr *graph = &graphs[i];
            int nvertice = graph->data[0];
            if (nvertice > maxvertice) {
                maxvertice = nvertice;
            }
            if (nvertice < minvertice) {
                minvertice = nvertice;
            }
            totalvertices = totalvertices + nvertice;
        }

        //https://devtalk.nvidia.com/default/topic/1026825/how-to-choose-how-many-threads-blocks-to-have-/
        //Therefore very small block sizes (e.g. 32 threads per block) may limit performance due to occupancy. 
        //Very large block sizes for example 1024 threads per block, may also limit performance, if there are resource limits 
        //(e.g. registers per thread usage, or shared memory usage) which prevent 2 threadblocks (in this example of 1024 threads per block) from being resident on a SM
        //Threadblock size choices in the range of 128 - 512 are less likely to run into the aforementioned issues


        int numblocks = totalvertices / BLOCK_SIZE_OPTIMAL;
        if ((totalvertices % BLOCK_SIZE_OPTIMAL) > 0) {
            numblocks++;
        }
        int nthreads = BLOCK_SIZE_OPTIMAL / BLOCK_FACTOR_OPTIMAL;

        int maxgraphsbyblock = BLOCK_SIZE_OPTIMAL / minvertice;
        if ((BLOCK_SIZE_OPTIMAL % minvertice) > 0) {
            maxgraphsbyblock++;
        }

        kernelAproxHullNumberGraphByBlockOptimal << <numblocks, nthreads, maxgraphsbyblock * sizeof (int)>>>(idxGraphsGpu, graphsGpu, cont, dataGraphsGpu, resultGpu, minvertice, maxvertice, totalvertices, maxgraphsbyblock);
        //        r = hipDeviceSynchronize();


        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("Time for the kernel(graph by block optimal): %2.f ms\n", milliseconds);

        if (verbose)
            printf("Read Result\n");


        if (r != hipSuccess) {
            fprintf(stderr, "Failed hipDeviceSynchronize \nError: %s\n", hipGetErrorString(r));
            exit(EXIT_FAILURE);
        }


        r = hipMemcpy(resultLocal, resultGpu, sizeof (int)*cont, hipMemcpyDeviceToHost);
        if (r != hipSuccess) {
            fprintf(stderr, "Failed to copy memory 4 \nError: %s\n", hipGetErrorString(r));
            exit(EXIT_FAILURE);
        }

        if (printResults)
            for (int i = 0; i < cont; i++) {
                printf("MinHullNumberAprox-Block Graph  optimal-%d: %d\n", i, resultLocal[i]);
            }
    }



    r = hipGetLastError();
    if (r != hipSuccess) {
        fprintf(stderr, "Failed in kernelAproxHullNumber \nError: %s\n", hipGetErrorString(r));
        exit(EXIT_FAILURE);
    }


    free(streams);
    free(resultLocal);
    free(graphsHost);
    free(idxGraphsGpu);
    hipFree(resultGpu);
    hipFree(graphsGpu);
    hipFree(dataGraphsGpu);
}

int serialAproxHullNumber(int *graphData) {
    int nvertices = graphData[0];
    //    printf("Nvertices: %d\n", nvertices);
    unsigned char *aux = (unsigned char *) malloc(sizeof (unsigned char)*nvertices);
    unsigned char *auxb = (unsigned char *) malloc(sizeof (unsigned char)*nvertices);
    if (aux == NULL || auxb == NULL) perror("Not memory allocation");
    int minHullSet = nvertices;

    for (int v = 0; v < nvertices; v++) {
        for (int j = 0; j < nvertices; j++) {
            aux[j] = 0;
        }

        int sizeHs = addVertToS(v, aux, graphData);
        int sSize = 1;
        int bv;
        do {
            bv = -1;
            int maiorGrau = 0;
            int maiorDeltaHs = 0;
            for (int i = 0; i < nvertices; i++) {
                if (aux[i] >= INCLUDED) {
                    continue;
                }
                COPY_ARRAY(aux, auxb, nvertices);
                int deltaHsi = addVertToS(i, auxb, graphData);

                int neighborCount = 0;
                for (int j = 0; j < nvertices; j++) {
                    if (auxb[j] == INCLUDED) {
                        neighborCount++;
                    }
                }

                if (bv == -1 || (deltaHsi >= maiorDeltaHs && neighborCount > maiorGrau)) {
                    maiorDeltaHs = deltaHsi;
                    maiorGrau = neighborCount;
                    bv = i;
                }
            }
            sizeHs = sizeHs + addVertToS(bv, aux, graphData);
            sSize++;
        } while (sizeHs < nvertices && sSize < minHullSet);
        minHullSet = MIN(minHullSet, sSize);
    }
    free(aux);
    free(auxb);
    return minHullSet;
}

void serialAproxHullNumberGraphs(graphCsr *graphs, int cont) {
    if (verbose) printf("Serial execution \n");
    int *results = new int[cont];
    clock_t begin_serial_time, end_serial_time;
    begin_serial_time = clock();
    for (int i = 0; i < cont; i++) {
        results[i] = serialAproxHullNumber(graphs[i].data);
        //        printf("MinHullNumberAprox-Serial Graph-%d: %d\n", i, results[i]);
    }
    end_serial_time = clock();

    double diff = end_serial_time - begin_serial_time;
    diff = diff / (CLOCKS_PER_SEC / 1000);

    printf("Time for the serial: %2.f ms\n", diff);


    if (printResults)
        for (int i = 0; i < cont; i++) {
            printf("MinHullNumberAprox-Serial Graph-%d: %d\n", i, results[i]);
        }

    free(results);
}

void processFiles(int argc, char** argv) {
    graphCsr* graphs = (graphCsr*) malloc((sizeof (graphCsr)) * argc);
    std::vector<int> values;

    int contGraph = 0;
    int verticesMedian = 0;

    for (int x = 1; x < argc; x++) {
        char* strFile = argv[x];
        DIR *dpdf;
        struct dirent *epdf;
        struct stat filestat;

        if (strFile[0] == '-') {
            continue;
        }

        dpdf = opendir(strFile);
        std::string filepath = std::string(strFile);

        while (dpdf && (epdf = readdir(dpdf))) {
            filepath = std::string(strFile) + "/" + epdf->d_name;
            if (epdf->d_name == "." || epdf->d_name == "..")
                continue;
            if (stat(filepath.c_str(), &filestat))
                continue;
            if (S_ISDIR(filestat.st_mode))
                continue;
            else break;
        }
        closedir(dpdf);

        std::string line, strCArray, strRArray;
        std::ifstream infile(filepath.c_str());

        if (infile) {
            while (getline(infile, line)) {
                if (line.at(0) != CHARACTER_INIT_COMMENT && line.at(1) != CHARACTER_INIT_COMMENT) {
                    if (strCArray.empty()) {
                        strCArray = line;
                    } else if (strRArray.empty()) {
                        strRArray = line;
                    }
                }
            }
            infile.close();
        } else {
            continue;
        }

        if (strCArray.empty() || strRArray.empty()) {
            perror("Invalid file format");
            continue;
        }

        std::stringstream stream(strCArray.c_str());
        values.clear();

        int n;
        while (stream >> n) {
            values.push_back(n);
        }
        strCArray.clear();

        int numVertices = values.size() - 1;
        stream.str("");

        std::stringstream stream2(strRArray);
        while (stream2 >> n) {
            values.push_back(n);
        }
        stream2.str("");
        strRArray.clear();

        int numedges = values.size() - (numVertices + 1);
        values.insert(values.begin(), numedges);
        values.insert(values.begin(), numVertices);
        int *data = new int[values.size()];
        std::copy(values.begin(), values.end(), data);

        values.clear();

        if (numVertices < 1) {
            continue;
        }

        graphCsr* graph = &graphs[contGraph];
        graph->data = data;
        contGraph++;
        verticesMedian = verticesMedian + numVertices;
    }

    if (contGraph > 0)
        printf("Processing: %d graphs  %dv/g (avg) \n", contGraph, verticesMedian / contGraph);

    if (graphByBlock || graphByThread || verticesBythread)
        parallelAproxHullNumberGraphs(graphs, contGraph);

    if (serial)
        serialAproxHullNumberGraphs(graphs, contGraph);

    for (int i = 0; i < contGraph; i++) {
        free(graphs[i].data);
    }
    free(graphs);
}

void runTest() {
    //    int numVertices = 10;
    //    int colIdx[] = {0, 3, 6, 9, 12, 15, 18, 21, 24, 27, 30};
    //    int sizeRowOffset = numVertices + 1;
    //    int rowOffset[] = {2, 5, 6, 3, 4, 6, 0, 4, 7, 1, 5, 7, 1, 2, 9, 0, 3, 9, 0, 1, 8, 2, 3, 8, 6, 7, 9, 4, 5, 8};

    printf("graphHullNumberAprox -[svtbnx] dir\n");

    int data[] = {10, 30,
        0, 3, 6, 9, 12, 15, 18, 21, 24, 27, 30,
        2, 5, 6, 3, 4, 6, 0, 4, 7, 1, 5, 7, 1, 2, 9, 0, 3, 9, 0, 1, 8, 2, 3, 8, 6, 7, 9, 4, 5, 8};
    graphCsr* graph = (graphCsr*) malloc(sizeof (graphCsr));
    graph->data = data;
    parallelAproxHullNumberGraphs(graph, 1);
}

int main(int argc, char** argv) {
    printf("Main\n");

    long opt = 0;

    while ((opt = getopt(argc, argv, "svtbnxo")) != -1) {
        switch (opt) {
            case 't':
                graphByThread = true;
                break;
            case 'b':
                graphByBlock = true;
                break;
            case 's':
                serial = true;
                break;
            case 'v':
                verbose = true;
                break;
            case 'x':
                verticesBythread = true;
                break;
            case 'o':
                graphByBlockOptimal = true;
                break;
            case 'n':
                printResults = false;
                break;
            case '?':
                printf("Unknow option: %c", char(opt));
                break;
        }
    }

    if (argc > 1) {
        processFiles(argc, argv);
    } else {
        runTest();
    }
    return 0;
}