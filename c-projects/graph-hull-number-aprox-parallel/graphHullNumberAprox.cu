#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <string>
#include <sstream>
#include <dirent.h>
#include <unistd.h>
#include <fstream>
#include <sys/stat.h>
#include <sys/types.h>
#include <vector>
#include <math.h> 
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include "nvgraph.h"

#define CHARACTER_INIT_COMMENT '#'

#define DEFAULT_THREAD_PER_BLOCK 28
#define DEFAULT_BLOCK 256 
#define MAX_DEFAULT_SIZE_QUEUE 256

#define WARP_SIZE 32
#define SINCLUDED 4
#define PROCESSED 3
#define INCLUDED 2
#define NEIGHBOOR_COUNT_INCLUDED 1
/* */
#define MAX(x, y) (((x) > (y)) ? (x) : (y))
#define MIN(x, y) (((x) < (y)) ? (x) : (y))
#define COPY_ARRAY(SRC,DST,LEN) { memcpy((DST), (SRC), LEN); }

#define verboseKernel false

bool verbose = false;
bool graphByThread = false;
bool graphByKernel = false;
bool verticesBythread = false;
bool serial = false;
bool printResults = true;

struct graphCsr {
    int *data;
    //    data[0] nVertices;
    //    data[0] nEdges
    //    data[2] *csrColIdxs;
    //    data[nVertices+3] *csrRowOffset;
};

__host__ __device__
int addVertToS(int vert, unsigned char* aux, int *graphData) {
    int countIncluded = 0;
    int nvertices = graphData[0];
    int *csrColIdxs = &graphData[2];
    int *csrRowOffset = &graphData[nvertices + 3];

    if (aux[vert] >= INCLUDED) {
        return countIncluded;
    }
    int headQueue = vert;
    int tailQueue = vert;
    aux[vert] = INCLUDED;

    while (headQueue <= tailQueue) {
        int verti = headQueue;
        if (verti >= nvertices || aux[verti] != INCLUDED) {
            headQueue++;
            continue;
        }
        int end = csrColIdxs[verti + 1];
        for (int j = csrColIdxs[verti]; j < end; j++) {
            int vertn = csrRowOffset[j];
            if (vertn >= nvertices) continue;
            if (vertn != verti && aux[vertn] < INCLUDED) {
                aux[vertn] = aux[vertn] + NEIGHBOOR_COUNT_INCLUDED;
                if (aux[vertn] == INCLUDED) {
                    headQueue = MIN(headQueue, vertn);
                    tailQueue = MAX(tailQueue, vertn);
                }
            }
        }
        aux[verti] = PROCESSED;
        countIncluded++;
    }
    aux[vert] = SINCLUDED;
    return countIncluded;
}

__host__ __device__
int exapandHullSetFromV(int v, int nvertices, unsigned char *aux, unsigned char *auxb, int *graphData, int idx) {
    for (int j = 0; j < nvertices; j++) {
        aux[j] = 0;
    }
    if (verboseKernel) printf("thread-%d: cleanded aux\n", idx);
    if (verboseKernel) printf("thread-%d: add Vert %d to S\n", idx, v);

    int sizeHs = addVertToS(v, aux, graphData);
    int sSize = 1;
    int bv;
    do {
        bv = -1;
        int maiorGrau = 0;
        int maiorDeltaHs = 0;
        for (int i = 0; i < nvertices; i++) {
            if (aux[i] >= INCLUDED) {
                continue;
            }
            COPY_ARRAY(aux, auxb, nvertices);
            int deltaHsi = addVertToS(i, auxb, graphData);

            int neighborCount = 0;
            for (int j = 0; j < nvertices; j++) {
                if (auxb[j] == INCLUDED) {
                    neighborCount++;
                }
            }

            if (bv == -1 || (deltaHsi >= maiorDeltaHs && neighborCount > maiorGrau)) {
                maiorDeltaHs = deltaHsi;
                maiorGrau = neighborCount;
                bv = i;
            }
        }
        if (verboseKernel) printf("thread-%d: add Vert %d to S\n", idx, bv);
        sizeHs = sizeHs + addVertToS(bv, aux, graphData);
        sSize++;
    } while (sizeHs < nvertices);
    return sSize;
}

__global__
void kernelAproxHullNumberIndexed(int offset, int* graphsGpu, int* dataGraphs, int* results) {
    //void kernelAproxHullNumber(graphCsr *graphs, int* results) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (verboseKernel) printf("thread-%d\n", idx);

    int start = graphsGpu[offset];
    if (verboseKernel) printf("thread-%d Graph-start: %d\n", idx, start);

    int *graphData = &dataGraphs[start];
    int nvertices = graphData[0];

    if (idx == 0) {
        results[offset] = nvertices;
    }
    __syncthreads();

    if (verboseKernel) printf("thread-%d Graph-nvertices: %d\n", idx, nvertices);
    if (verboseKernel) printf("thread-%d Graph-edges: %d\n", idx, graphData[1]);

    unsigned char *aux = new unsigned char [nvertices * 2];
    unsigned char *auxb = &aux[nvertices];
    int minHullSet = nvertices;
    int v = idx;

    int sSize = exapandHullSetFromV(v, nvertices, aux, auxb, graphData, idx);

    minHullSet = MIN(minHullSet, sSize);

    if (verboseKernel) printf("thread-%d: minHullSet %d\n", idx, minHullSet);

    free(aux);
    //    free(auxb);
    if (verboseKernel) printf("thread-%d: memory freed\n", idx);

    atomicMin(&results[offset], minHullSet);
}

__global__
void kernelAproxHullNumberByVertex(int offset, int* graphsGpu,
        int* dataGraphs, int* results) {
    //void kernelAproxHullNumber(graphCsr *graphs, int* results) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (verboseKernel)
        printf("thread-%d\n", idx);

    int start = graphsGpu[offset];
    if (verboseKernel)
        printf("thread-%d Graph-start: %d\n", idx, start);

    int *graphData = &dataGraphs[start];
    int nvertices = graphData[0];

    if (idx == 0) {
        results[offset] = nvertices;
    }
    __syncthreads();

    if (idx < nvertices) {
        if (verboseKernel)
            printf("thread-%d Graph-nvertices: %d\n", idx, nvertices);
        if (verboseKernel)
            printf("thread-%d Graph-edges: %d\n", idx, graphData[1]);

        unsigned char *aux = new unsigned char [nvertices * 2];
        unsigned char *auxb = &aux[nvertices];
        int minHullSet = nvertices;
        int v = idx;

        int sSize = exapandHullSetFromV(v, nvertices, aux, auxb, graphData, idx);
        minHullSet = MIN(minHullSet, sSize);

        if (verboseKernel)
            printf("thread-%d: minHullSet %d\n", idx, minHullSet);
        if (verboseKernel)
            printf("thread-%d: memory freed\n", idx);
        atomicMin(&results[offset], minHullSet);
        free(aux);
    }
}

__global__
void kernelAproxHullNumber(int* graphsGpu, int* dataGraphs, int* results) {
    //void kernelAproxHullNumber(graphCsr *graphs, int* results) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (verboseKernel)
        printf("thread-%d\n", idx);

    int start = graphsGpu[idx];
    if (verboseKernel)
        printf("thread-%d Graph-start: %d\n", idx, start);

    int *graphData = &dataGraphs[start];
    int nvertices = graphData[0];

    if (verboseKernel)
        printf("thread-%d Graph-nvertices: %d\n", idx, nvertices);
    if (verboseKernel)
        printf("thread-%d Graph-edges: %d\n", idx, graphData[1]);

    unsigned char *aux = new unsigned char [nvertices * 2];
    //    unsigned char *auxb = new unsigned char [nvertices];
    unsigned char *auxb = &aux[nvertices];

    int minHullSet = nvertices;

    for (int v = 0; v < nvertices; v++) {
        for (int j = 0; j < nvertices; j++) {
            aux[j] = 0;
        }
        if (verboseKernel) printf("thread-%d: cleanded aux\n", idx);
        if (verboseKernel) printf("thread-%d: add Vert %d to S\n", idx, v);

        int sizeHs = addVertToS(v, aux, graphData);
        int sSize = 1;
        int bv;
        do {
            bv = -1;
            int maiorGrau = 0;
            int maiorDeltaHs = 0;
            for (int i = 0; i < nvertices; i++) {
                if (aux[i] >= INCLUDED) {
                    continue;
                }
                COPY_ARRAY(aux, auxb, nvertices);
                int deltaHsi = addVertToS(i, auxb, graphData);

                int neighborCount = 0;
                for (int j = 0; j < nvertices; j++) {
                    if (auxb[j] == INCLUDED) {
                        neighborCount++;
                    }
                }

                if (bv == -1 || (deltaHsi >= maiorDeltaHs && neighborCount > maiorGrau)) {
                    maiorDeltaHs = deltaHsi;
                    maiorGrau = neighborCount;
                    bv = i;
                }
            }
            if (verboseKernel) printf("thread-%d: add Vert %d to S\n", idx, bv);
            sizeHs = sizeHs + addVertToS(bv, aux, graphData);
            sSize++;
        } while (sizeHs < nvertices);
        minHullSet = MIN(minHullSet, sSize);
        if (verboseKernel) printf("thread-%d: minHullSet %d\n", idx, minHullSet);
    }
    free(aux);
    //    free(auxb);
    if (verboseKernel) printf("thread-%d: memory freed\n", idx);
    results[idx] = minHullSet;
}

int parallelAproxHullNumberGraphs(graphCsr *graphs, int cont) {
    if (verbose) printf("ParallelAproxHullNumberGraphs\n");
    int numbytesDataGraph = 0;
    hipError_t r;
    for (int i = 0; i < cont; i++) {
        int size = (graphs[i].data[0] + 3 + graphs[i].data[1]) * sizeof (int);
        numbytesDataGraph = numbytesDataGraph + size;
    }

    if (verbose) printf("Cuda Malloc Graph\n");

    int device;
    hipDeviceProp_t deviceProp;
    hipGetDevice(&device);
    hipGetDeviceProperties(&deviceProp, device);

    if ((deviceProp.concurrentKernels == 0)) {
        perror("not support for concurrent kernel\n");
    }
    if (verbose)
        printf("Compute SM %d.%d with %d multi-processors\n", deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount);

    int* dataGraphsGpu;
    int *graphsGpu;
    int* resultGpu;
    int* graphsHost = new int[cont];
    hipMalloc((void**) &resultGpu, cont * sizeof (int));
    hipMalloc((void**) &dataGraphsGpu, numbytesDataGraph);
    hipMalloc((void**) &graphsGpu, cont * sizeof (int));
    r = hipMemcpy(graphsGpu, graphs, cont * sizeof (int), hipMemcpyHostToDevice);
    int offset = 0;

    //if (verbose) printf("Cuda Atrib Graph\n");
    for (int i = 0; i < cont; i++) {
        //if (verbose) printf("Cuda Atrib Graph-%d\n", i);
        graphCsr *graph = &graphs[i];
        int sizeGraph = graph->data[0] + 3 + graph->data[1];
        int nbytes = sizeGraph * sizeof (int);
        r = hipMemcpy(dataGraphsGpu + offset, graph->data, nbytes, hipMemcpyHostToDevice);
        if (r != hipSuccess) {
            fprintf(stderr, "Failed to copy memory 1 \nError: %s\n", hipGetErrorString(r));
            exit(EXIT_FAILURE);
        }
        graphsHost[i] = offset;
        offset = offset + sizeGraph;
    }

    r = hipMemcpy(graphsGpu, graphsHost, cont * sizeof (int), hipMemcpyHostToDevice);
    if (r != hipSuccess) {
        fprintf(stderr, "Failed to copy memory 2 \nError: %s\n", hipGetErrorString(r));
        exit(EXIT_FAILURE);
    }

    hipStream_t *streams = (hipStream_t *) malloc(cont * sizeof (hipStream_t));
    for (int i = 0; i < cont; i++) {
        hipStreamCreate(&(streams[i]));
    }

    if (verbose) printf("Launch Kernel\n");

    int* resultLocal = new int[cont];

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    /* Graphs by Thread */
    if (graphByThread) {
        if (verbose) printf("Lanch Kernel One graph by thread\n");

        hipEventRecord(start);

        kernelAproxHullNumber << <1, cont>>>(graphsGpu, dataGraphsGpu, resultGpu);
        r = hipDeviceSynchronize();

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("Time for the kernel(one graph by thread): %2.f ms\n", milliseconds);

        if (verbose) printf("Read Result\n");

        if (r != hipSuccess) {
            fprintf(stderr, "Failed hipDeviceSynchronize \nError: %s\n", hipGetErrorString(r));
            exit(EXIT_FAILURE);
        }

        r = hipMemcpy(resultLocal, resultGpu, sizeof (int)*cont, hipMemcpyDeviceToHost);
        if (r != hipSuccess) {
            fprintf(stderr, "Failed to copy memory 4 \nError: %s\n", hipGetErrorString(r));
            exit(EXIT_FAILURE);
        }

        if (printResults)
            for (int i = 0; i < cont; i++) {
                printf("MinHullNumberAprox-Thread Graph-%d: %d\n", i, resultLocal[i]);
            }
    }

    /* Graphs by Block (Kernel) */
    if (graphByKernel) {
        if (verbose)
            printf("Lauch Kernel One graph by kernel\n");
        hipEventRecord(start);

        for (int i = 0; i < cont; i++) {
            graphCsr *graph = &graphs[i];
            int nvertice = graph->data[0];
            kernelAproxHullNumberIndexed << <1, nvertice, 0, streams[i]>>>(i, graphsGpu, dataGraphsGpu, resultGpu);
        }
        //        r = hipDeviceSynchronize();

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("Time for the kernel(one graph by block): %2.f ms\n", milliseconds);

        if (verbose)
            printf("Read Result\n");


        if (r != hipSuccess) {
            fprintf(stderr, "Failed hipDeviceSynchronize \nError: %s\n", hipGetErrorString(r));
            exit(EXIT_FAILURE);
        }


        r = hipMemcpy(resultLocal, resultGpu, sizeof (int)*cont, hipMemcpyDeviceToHost);
        if (r != hipSuccess) {
            fprintf(stderr, "Failed to copy memory 4 \nError: %s\n", hipGetErrorString(r));
            exit(EXIT_FAILURE);
        }

        if (printResults)
            for (int i = 0; i < cont; i++) {
                printf("MinHullNumberAprox-Block Graph-%d: %d\n", i, resultLocal[i]);
            }
    }

    /* Vertices by Thread */
    if (verticesBythread) {
        if (verbose)
            printf("Vertices by Thread One graph by kernel\n");
        hipEventRecord(start);

        for (int i = 0; i < cont; i++) {
            graphCsr *graph = &graphs[i];
            int nvertice = graph->data[0];
            int nblocks = nvertice / DEFAULT_THREAD_PER_BLOCK;
            if ((nvertice % DEFAULT_THREAD_PER_BLOCK) > 0) {
                nblocks++;
            }
            //            int nthreads = MIN(DEFAULT_THREAD_PER_BLOCK, nvertice);
            int nthreads = DEFAULT_THREAD_PER_BLOCK;
            kernelAproxHullNumberByVertex << <nblocks, nthreads, 0, streams[i]>>>(i, graphsGpu, dataGraphsGpu, resultGpu);
        }
        //        r = hipDeviceSynchronize();

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("Time for Vt/Thread kernel(one graph by kernel): %2.f ms\n", milliseconds);

        if (verbose)
            printf("Read Result\n");


        if (r != hipSuccess) {
            fprintf(stderr, "Failed hipDeviceSynchronize \nError: %s\n", hipGetErrorString(r));
            exit(EXIT_FAILURE);
        }


        r = hipMemcpy(resultLocal, resultGpu, sizeof (int)*cont, hipMemcpyDeviceToHost);
        if (r != hipSuccess) {
            fprintf(stderr, "Failed to copy memory 4 \nError: %s\n", hipGetErrorString(r));
            exit(EXIT_FAILURE);
        }

        if (printResults)
            for (int i = 0; i < cont; i++) {
                printf("MinHullNumberAprox-Vertices Thread-%d: %d\n", i, resultLocal[i]);
            }
    }



    r = hipGetLastError();
    if (r != hipSuccess) {
        fprintf(stderr, "Failed in kernelAproxHullNumber \nError: %s\n", hipGetErrorString(r));
        exit(EXIT_FAILURE);
    }


    free(streams);
    free(resultLocal);
    free(graphsHost);
    hipFree(resultGpu);
    hipFree(graphsGpu);
    hipFree(dataGraphsGpu);
}

int serialAproxHullNumber(int *graphData) {
    int nvertices = graphData[0];
    //    printf("Nvertices: %d\n", nvertices);
    unsigned char *aux = (unsigned char *) malloc(sizeof (unsigned char)*nvertices);
    unsigned char *auxb = (unsigned char *) malloc(sizeof (unsigned char)*nvertices);
    if (aux == NULL || auxb == NULL) perror("Not memory allocation");
    int minHullSet = nvertices;

    for (int v = 0; v < nvertices; v++) {
        for (int j = 0; j < nvertices; j++) {
            aux[j] = 0;
        }

        int sizeHs = addVertToS(v, aux, graphData);
        int sSize = 1;
        int bv;
        do {
            bv = -1;
            int maiorGrau = 0;
            int maiorDeltaHs = 0;
            for (int i = 0; i < nvertices; i++) {
                if (aux[i] >= INCLUDED) {
                    continue;
                }
                COPY_ARRAY(aux, auxb, nvertices);
                int deltaHsi = addVertToS(i, auxb, graphData);

                int neighborCount = 0;
                for (int j = 0; j < nvertices; j++) {
                    if (auxb[j] == INCLUDED) {
                        neighborCount++;
                    }
                }

                if (bv == -1 || (deltaHsi >= maiorDeltaHs && neighborCount > maiorGrau)) {
                    maiorDeltaHs = deltaHsi;
                    maiorGrau = neighborCount;
                    bv = i;
                }
            }
            sizeHs = sizeHs + addVertToS(bv, aux, graphData);
            sSize++;
        } while (sizeHs < nvertices);
        minHullSet = MIN(minHullSet, sSize);
    }
    free(aux);
    free(auxb);
    return minHullSet;
}

void serialAproxHullNumberGraphs(graphCsr *graphs, int cont) {
    if (verbose) printf("Serial execution \n");
    int *results = new int[cont];
    clock_t begin_serial_time, end_serial_time;
    begin_serial_time = clock();
    for (int i = 0; i < cont; i++) {
        results[i] = serialAproxHullNumber(graphs[i].data);
        //        printf("MinHullNumberAprox-Serial Graph-%d: %d\n", i, results[i]);
    }
    end_serial_time = clock();

    double diff = end_serial_time - begin_serial_time;
    diff = diff / (CLOCKS_PER_SEC / 1000);

    printf("Time for the serial: %2.f ms\n", diff);


    if (printResults)
        for (int i = 0; i < cont; i++) {
            printf("MinHullNumberAprox-Serial Graph-%d: %d\n", i, results[i]);
        }

    free(results);
}

void processFiles(int argc, char** argv) {
    graphCsr* graphs = (graphCsr*) malloc((sizeof (graphCsr)) * argc);
    std::vector<int> values;

    int contGraph = 0;
    int verticesMedian = 0;

    for (int x = 1; x < argc; x++) {
        char* strFile = argv[x];
        DIR *dpdf;
        struct dirent *epdf;
        struct stat filestat;

        if (strFile[0] == '-') {
            continue;
        }

        dpdf = opendir(strFile);
        std::string filepath = std::string(strFile);

        while (dpdf && (epdf = readdir(dpdf))) {
            filepath = std::string(strFile) + "/" + epdf->d_name;
            if (epdf->d_name == "." || epdf->d_name == "..")
                continue;
            if (stat(filepath.c_str(), &filestat))
                continue;
            if (S_ISDIR(filestat.st_mode))
                continue;
            else break;
        }
        closedir(dpdf);

        std::string line, strCArray, strRArray;
        std::ifstream infile(filepath.c_str());

        if (infile) {
            while (getline(infile, line)) {
                if (line.at(0) != CHARACTER_INIT_COMMENT && line.at(1) != CHARACTER_INIT_COMMENT) {
                    if (strCArray.empty()) {
                        strCArray = line;
                    } else if (strRArray.empty()) {
                        strRArray = line;
                    }
                }
            }
            infile.close();
        } else {
            continue;
        }

        if (strCArray.empty() || strRArray.empty()) {
            perror("Invalid file format");
            continue;
        }

        std::stringstream stream(strCArray.c_str());
        values.clear();

        int n;
        while (stream >> n) {
            values.push_back(n);
        }
        strCArray.clear();

        int numVertices = values.size() - 1;
        stream.str("");

        std::stringstream stream2(strRArray);
        while (stream2 >> n) {
            values.push_back(n);
        }
        stream2.str("");
        strRArray.clear();

        int numedges = values.size() - (numVertices + 1);
        values.insert(values.begin(), numedges);
        values.insert(values.begin(), numVertices);
        int *data = new int[values.size()];
        std::copy(values.begin(), values.end(), data);

        values.clear();

        if (numVertices < 1) {
            continue;
        }

        graphCsr* graph = &graphs[contGraph];
        graph->data = data;
        contGraph++;
        verticesMedian = verticesMedian + numVertices;
    }

    if (contGraph > 0)
        printf("Processing: %d graphs  %dv/g (avg) \n", contGraph, verticesMedian / contGraph);

    if (graphByKernel || graphByThread || verticesBythread)
        parallelAproxHullNumberGraphs(graphs, contGraph);

    if (serial)
        serialAproxHullNumberGraphs(graphs, contGraph);

    for (int i = 0; i < contGraph; i++) {
        free(graphs[i].data);
    }
    free(graphs);
}

void runTest() {
    //    int numVertices = 10;
    //    int colIdx[] = {0, 3, 6, 9, 12, 15, 18, 21, 24, 27, 30};
    //    int sizeRowOffset = numVertices + 1;
    //    int rowOffset[] = {2, 5, 6, 3, 4, 6, 0, 4, 7, 1, 5, 7, 1, 2, 9, 0, 3, 9, 0, 1, 8, 2, 3, 8, 6, 7, 9, 4, 5, 8};

    printf("graphHullNumberAprox -[svtbnx] dir\n");

    int data[] = {10, 30,
        0, 3, 6, 9, 12, 15, 18, 21, 24, 27, 30,
        2, 5, 6, 3, 4, 6, 0, 4, 7, 1, 5, 7, 1, 2, 9, 0, 3, 9, 0, 1, 8, 2, 3, 8, 6, 7, 9, 4, 5, 8};
    graphCsr* graph = (graphCsr*) malloc(sizeof (graphCsr));
    graph->data = data;
    parallelAproxHullNumberGraphs(graph, 1);
}

int main(int argc, char** argv) {
    printf("Main\n");

    long opt = 0;

    while ((opt = getopt(argc, argv, "svtbnx")) != -1) {
        switch (opt) {
            case 't':
                graphByThread = true;
                break;
            case 'b':
                graphByKernel = true;
                break;
            case 's':
                serial = true;
                break;
            case 'v':
                verbose = true;
                break;
            case 'x':
                verticesBythread = true;
                break;
            case 'n':
                printResults = false;
                break;
            case '?':
                printf("Unknow option: %c", char(opt));
                break;
        }
    }

    if (argc > 1) {
        processFiles(argc, argv);
    } else {
        runTest();
    }
    return 0;
}