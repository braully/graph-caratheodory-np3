#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <string>
#include <sstream>
#include <dirent.h>
#include <unistd.h>
#include <fstream>
#include <sys/stat.h>
#include <sys/types.h>
#include <vector>
#include <math.h> 
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "nvgraph.h"

#define CHARACTER_INIT_COMMENT '#'

#define DEFAULT_THREAD_PER_BLOCK 256
#define DEFAULT_BLOCK 256 
#define MAX_DEFAULT_SIZE_QUEUE 256

#define SINCLUDED 4
#define PROCESSED 3
#define INCLUDED 2
#define NEIGHBOOR_COUNT_INCLUDED 1
/* */
#define MAX(x, y) (((x) > (y)) ? (x) : (y))
#define MIN(x, y) (((x) < (y)) ? (x) : (y))
#define COPY_ARRAY(SRC,DST,LEN) { memcpy((DST), (SRC), LEN); }

#define verboseKernel false
bool verbose = true;
bool graphByThread = false;
bool graphByKernel = false;
bool serial = false;

struct graphCsr {
    int *data;
    //    data[0] nVertices;
    //    data[0] nEdges
    //    data[2] *csrColIdxs;
    //    data[nVertices+3] *csrRowOffset;
};

__host__ __device__
int addVertToS(int vert, unsigned char* aux, int *graphData) {
    int countIncluded = 0;
    int nvertices = graphData[0];
    int *csrColIdxs = &graphData[2];
    int *csrRowOffset = &graphData[nvertices + 3];

    if (aux[vert] >= INCLUDED) {
        return countIncluded;
    }
    int headQueue = vert;
    int tailQueue = vert;
    aux[vert] = INCLUDED;

    while (headQueue <= tailQueue) {
        int verti = headQueue;
        if (verti >= nvertices || aux[verti] != INCLUDED) {
            headQueue++;
            continue;
        }
        int end = csrColIdxs[verti + 1];
        for (int j = csrColIdxs[verti]; j < end; j++) {
            int vertn = csrRowOffset[j];
            if (vertn >= nvertices) continue;
            if (vertn != verti && aux[vertn] < INCLUDED) {
                aux[vertn] = aux[vertn] + NEIGHBOOR_COUNT_INCLUDED;
                if (aux[vertn] == INCLUDED) {
                    headQueue = MIN(headQueue, vertn);
                    tailQueue = MAX(tailQueue, vertn);
                }
            }
        }
        aux[verti] = PROCESSED;
        countIncluded++;
    }
    aux[vert] = SINCLUDED;
    return countIncluded;
}

__host__ __device__
int exapandHullSetFromV(int v, int nvertices, unsigned char *aux, unsigned char *auxb, int *graphData, int idx) {
    for (int j = 0; j < nvertices; j++) {
        aux[j] = 0;
    }
    if (verboseKernel) printf("thread-%d: cleanded aux\n", idx);
    if (verboseKernel) printf("thread-%d: add Vert %d to S\n", idx, v);

    int sizeHs = addVertToS(v, aux, graphData);
    int sSize = 1;
    int bv;
    do {
        bv = -1;
        int maiorGrau = 0;
        int maiorDeltaHs = 0;
        for (int i = 0; i < nvertices; i++) {
            if (aux[i] >= INCLUDED) {
                continue;
            }
            COPY_ARRAY(aux, auxb, nvertices);
            int deltaHsi = addVertToS(i, auxb, graphData);

            int neighborCount = 0;
            for (int j = 0; j < nvertices; j++) {
                if (auxb[j] == INCLUDED) {
                    neighborCount++;
                }
            }

            if (bv == -1 || (deltaHsi >= maiorDeltaHs && neighborCount > maiorGrau)) {
                maiorDeltaHs = deltaHsi;
                maiorGrau = neighborCount;
                bv = i;
            }
        }
        if (verboseKernel) printf("thread-%d: add Vert %d to S\n", idx, bv);
        sizeHs = sizeHs + addVertToS(bv, aux, graphData);
        sSize++;
    } while (sizeHs < nvertices);
    return sSize;
}

__global__
void kernelSerialAproxHullNumberIndexed(int offset, int* graphsGpu, int* dataGraphs, int* results) {
    //void kernelSerialAproxHullNumber(graphCsr *graphs, int* results) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (verboseKernel) printf("thread-%d\n", idx);

    int start = graphsGpu[offset];
    if (verboseKernel) printf("thread-%d Graph-start: %d\n", idx, start);

    int *graphData = &dataGraphs[start];
    int nvertices = graphData[0];

    if (idx == 0) {
        results[offset] = nvertices;
    }
    __syncthreads();

    if (verboseKernel) printf("thread-%d Graph-nvertices: %d\n", idx, nvertices);
    if (verboseKernel) printf("thread-%d Graph-edges: %d\n", idx, graphData[1]);

    unsigned char *aux = new unsigned char [nvertices];
    unsigned char *auxb = new unsigned char [nvertices];
    int minHullSet = nvertices;
    int v = idx;

    int sSize = exapandHullSetFromV(v, nvertices, aux, auxb, graphData, idx);

    minHullSet = MIN(minHullSet, sSize);

    if (verboseKernel) printf("thread-%d: minHullSet %d\n", idx, minHullSet);

    free(aux);
    free(auxb);
    if (verboseKernel) printf("thread-%d: memory freed\n", idx);

    atomicMin(&results[offset], minHullSet);
}

__global__
void kernelSerialAproxHullNumber(int* graphsGpu, int* dataGraphs, int* results) {
    //void kernelSerialAproxHullNumber(graphCsr *graphs, int* results) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (verboseKernel) printf("thread-%d\n", idx);

    int start = graphsGpu[idx];
    if (verboseKernel) printf("thread-%d Graph-start: %d\n", idx, start);

    int *graphData = &dataGraphs[start];
    int nvertices = graphData[0];

    if (verboseKernel) printf("thread-%d Graph-nvertices: %d\n", idx, nvertices);
    if (verboseKernel) printf("thread-%d Graph-edges: %d\n", idx, graphData[1]);

    unsigned char *aux = new unsigned char [nvertices];
    unsigned char *auxb = new unsigned char [nvertices];
    int minHullSet = nvertices;

    for (int v = 0; v < nvertices; v++) {
        for (int j = 0; j < nvertices; j++) {
            aux[j] = 0;
        }
        if (verboseKernel) printf("thread-%d: cleanded aux\n", idx);
        if (verboseKernel) printf("thread-%d: add Vert %d to S\n", idx, v);

        int sizeHs = addVertToS(v, aux, graphData);
        int sSize = 1;
        int bv;
        do {
            bv = -1;
            int maiorGrau = 0;
            int maiorDeltaHs = 0;
            for (int i = 0; i < nvertices; i++) {
                if (aux[i] >= INCLUDED) {
                    continue;
                }
                COPY_ARRAY(aux, auxb, nvertices);
                int deltaHsi = addVertToS(i, auxb, graphData);

                int neighborCount = 0;
                for (int j = 0; j < nvertices; j++) {
                    if (auxb[j] == INCLUDED) {
                        neighborCount++;
                    }
                }

                if (bv == -1 || (deltaHsi >= maiorDeltaHs && neighborCount > maiorGrau)) {
                    maiorDeltaHs = deltaHsi;
                    maiorGrau = neighborCount;
                    bv = i;
                }
            }
            if (verboseKernel) printf("thread-%d: add Vert %d to S\n", idx, bv);
            sizeHs = sizeHs + addVertToS(bv, aux, graphData);
            sSize++;
        } while (sizeHs < nvertices);
        minHullSet = MIN(minHullSet, sSize);
        if (verboseKernel) printf("thread-%d: minHullSet %d\n", idx, minHullSet);
    }
    free(aux);
    free(auxb);
    if (verboseKernel) printf("thread-%d: memory freed\n", idx);
    results[idx] = minHullSet;
}

int serialAproxHullNumberGraphs(graphCsr *graphs, int cont) {
    if (verbose) printf("serialAproxHullNumberGraphs\n");
    int numbytesDataGraph = 0;
    hipError_t r;
    for (int i = 0; i < cont; i++) {
        int size = (graphs[i].data[0] + 3 + graphs[i].data[1]) * sizeof (int);
        numbytesDataGraph = numbytesDataGraph + size;
    }

    if (verbose) printf("Cuda Malloc Graph\n");

    int* dataGraphsGpu;
    int *graphsGpu;
    int* resultGpu;
    int* graphsHost = new int[cont];
    hipMalloc((void**) &resultGpu, cont * sizeof (int));
    hipMalloc((void**) &dataGraphsGpu, numbytesDataGraph);
    hipMalloc((void**) &graphsGpu, cont * sizeof (int));
    r = hipMemcpy(graphsGpu, graphs, cont * sizeof (graphCsr), hipMemcpyHostToDevice);

    int offset = 0;

    if (verbose) printf("Cuda Atrib Graph\n");
    for (int i = 0; i < cont; i++) {
        if (verbose) printf("Cuda Atrib Graph-%d\n", i);
        graphCsr *graph = &graphs[i];
        int sizeGraph = graph->data[0] + 3 + graph->data[1];
        int nbytes = sizeGraph * sizeof (int);
        r = hipMemcpy(dataGraphsGpu + offset, graph->data, nbytes, hipMemcpyHostToDevice);
        if (r != hipSuccess) {
            fprintf(stderr, "Failed to copy memory 1 \nError: %s\n", hipGetErrorString(r));
            exit(EXIT_FAILURE);
        }
        graphsHost[i] = offset;
        offset = offset + sizeGraph;
    }

    r = hipMemcpy(graphsGpu, graphsHost, cont * sizeof (int), hipMemcpyHostToDevice);
    if (r != hipSuccess) {
        fprintf(stderr, "Failed to copy memory 2 \nError: %s\n", hipGetErrorString(r));
        exit(EXIT_FAILURE);
    }

    if (verbose) printf("Launch Kernel\n");

    if (graphByThread) {
        if (verbose) printf("One graph by thread\n");
        kernelSerialAproxHullNumber << <1, cont>>>(graphsGpu, dataGraphsGpu, resultGpu);
    } else {
        if (verbose) printf("One graph by kernel\n");
        for (int i = 0; i < cont; i++) {
            graphCsr *graph = &graphs[i];
            int nvertice = graph->data[0];
            kernelSerialAproxHullNumberIndexed << <1, nvertice>>>(i, graphsGpu, dataGraphsGpu, resultGpu);
        }
    }

    if (verbose) printf("Read Result\n");

    r = hipDeviceSynchronize();
    if (r != hipSuccess) {
        fprintf(stderr, "Failed hipDeviceSynchronize \nError: %s\n", hipGetErrorString(r));
        exit(EXIT_FAILURE);
    }


    int* resultLocal = new int[cont];
    r = hipMemcpy(resultLocal, resultGpu, sizeof (int)*cont, hipMemcpyDeviceToHost);
    if (r != hipSuccess) {
        fprintf(stderr, "Failed to copy memory 4 \nError: %s\n", hipGetErrorString(r));
        exit(EXIT_FAILURE);
    }

    //    r = hipGetLastError();
    //    if (r != hipSuccess) {
    //        fprintf(stderr, "Failed in kernelSerialAproxHullNumber \nError: %s\n", hipGetErrorString(r));
    //        exit(EXIT_FAILURE);
    //    }

    for (int i = 0; i < cont; i++) {
        printf("MinHullNumberAprox Graph-%d: %d\n", i, resultLocal[i]);
    }

    free(resultLocal);
    free(graphsHost);
    hipFree(resultGpu);
    hipFree(graphsGpu);
    hipFree(dataGraphsGpu);
}

void processFiles(int argc, char** argv) {
    graphCsr* graphs = (graphCsr*) malloc((sizeof (graphCsr)) * argc);
    std::vector<int> values;

    int contGraph = 0;

    for (int x = 1; x < argc; x++) {
        char* strFile = "graph-test.txt";
        strFile = argv[x];
        DIR *dpdf;
        struct dirent *epdf;
        struct stat filestat;

        dpdf = opendir(strFile);
        std::string filepath = std::string(strFile);

        while (dpdf && (epdf = readdir(dpdf))) {
            filepath = std::string(strFile) + "/" + epdf->d_name;
            if (epdf->d_name == "." || epdf->d_name == "..")
                continue;
            if (stat(filepath.c_str(), &filestat))
                continue;
            if (S_ISDIR(filestat.st_mode))
                continue;
            else break;
        }
        closedir(dpdf);

        std::string line, strCArray, strRArray;
        std::ifstream infile(filepath.c_str());

        if (infile) {
            while (getline(infile, line)) {
                if (line.at(0) != CHARACTER_INIT_COMMENT) {
                    if (strCArray.empty()) {
                        strCArray = line;
                    } else if (strRArray.empty()) {
                        strRArray = line;
                    }
                }
            }
            infile.close();
        } else {
            continue;
        }

        if (strCArray.empty() || strRArray.empty()) {
            perror("Invalid file format");
            continue;
        }

        std::stringstream stream(strCArray.c_str());
        values.clear();

        int n;
        while (stream >> n) {
            values.push_back(n);
        }
        strCArray.clear();

        int numVertices = values.size() - 1;
        stream.str("");

        std::stringstream stream2(strRArray);
        while (stream2 >> n) {
            values.push_back(n);
        }
        stream2.str("");
        strRArray.clear();

        int numedges = values.size() - (numVertices + 1);
        values.insert(values.begin(), numedges);
        values.insert(values.begin(), numVertices);
        int *data = new int[values.size()];
        std::copy(values.begin(), values.end(), data);

        values.clear();

        graphCsr* graph = &graphs[contGraph];
        graph->data = data;
        contGraph++;
    }
    serialAproxHullNumberGraphs(graphs, contGraph);
}

void runTest() {
    //    int numVertices = 10;
    //    int colIdx[] = {0, 3, 6, 9, 12, 15, 18, 21, 24, 27, 30};
    //    int sizeRowOffset = numVertices + 1;
    //    int rowOffset[] = {2, 5, 6, 3, 4, 6, 0, 4, 7, 1, 5, 7, 1, 2, 9, 0, 3, 9, 0, 1, 8, 2, 3, 8, 6, 7, 9, 4, 5, 8};
    int data[] = {10, 30,
        0, 3, 6, 9, 12, 15, 18, 21, 24, 27, 30,
        2, 5, 6, 3, 4, 6, 0, 4, 7, 1, 5, 7, 1, 2, 9, 0, 3, 9, 0, 1, 8, 2, 3, 8, 6, 7, 9, 4, 5, 8};
    graphCsr* graph = (graphCsr*) malloc(sizeof (graphCsr));
    graph->data = data;
    //    graph->nVertices = numVertices;
    //    graph->csrColIdxs = colIdx;
    //    graph->csrRowOffset = rowOffset;
    //    int minSerialAprox = serialAproxHullNumber(graph);
    //    printf("MinAproxHullSet: %d\n", minSerialAprox);

    serialAproxHullNumberGraphs(graph, 1);
}

int main(int argc, char** argv) {
    printf("Main\n");
    if (argc > 1) {
        processFiles(argc, argv);
    } else {
        runTest();
    }
    return 0;
}