#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <string>
#include <sstream>
#include <dirent.h>
#include <unistd.h>
#include <fstream>
#include <sys/stat.h>
#include <sys/types.h>
#include <vector>
#include <math.h> 
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <limits.h>
#include "nvgraph.h"

#define CHARACTER_INIT_COMMENT '#'


#define BLOCK_WINDOWS 32
#define BLOCK_WINDOWS_TRUNK 128
#define BLOCK_SIZE_OPTIMAL 256
#define BLOCK_FACTOR_OPTIMAL 2

#define DEFAULT_THREAD_PER_BLOCK 32
#define DEFAULT_BLOCK 256 
#define MAX_DEFAULT_SIZE_QUEUE 256

#define WARP_SIZE 32
#define SINCLUDED 4
#define PROCESSED 3
#define INCLUDED 2
#define NEIGHBOOR_COUNT_INCLUDED 1
/* */
#define MAX(x, y) (((x) > (y)) ? (x) : (y))
#define MIN(x, y) (((x) < (y)) ? (x) : (y))
#define COPY_ARRAY(SRC,DST,LEN) { memcpy((DST), (SRC), LEN); }

#define verboseKernel false

bool verbose = false;
bool graphByThread = false;
bool graphByBlock = false;
bool graphByBlockOptimal = false;
bool verticesBythread = false;
bool serial = false;
bool printResults = true;

struct graphCsr {
    int *data;
    //    data[0] nVertices;
    //    data[0] nEdges
    //    data[2] *csrColIdxs;
    //    data[nVertices+3] *csrRowOffset;
};

int compare(const void *s1, const void *s2) {
    struct graphCsr *g1 = (struct graphCsr *) s1;
    struct graphCsr *g2 = (struct graphCsr *) s2;
    int cmp = (g2->data[0] - g1->data[0]);
    return cmp;
}

__host__ __device__
int addVertToS(int vert, unsigned char* aux, int *graphData) {
    int countIncluded = 0;
    int nvertices = graphData[0];
    int *csrColIdxs = &graphData[2];
    int *csrRowOffset = &graphData[nvertices + 3];

    if (aux[vert] >= INCLUDED) {
        return countIncluded;
    }
    int headQueue = vert;
    int tailQueue = vert;
    aux[vert] = INCLUDED;

    while (headQueue <= tailQueue) {
        int verti = headQueue;
        if (verti >= nvertices || aux[verti] != INCLUDED) {
            headQueue++;
            continue;
        }
        int end = csrColIdxs[verti + 1];
        for (int j = csrColIdxs[verti]; j < end; j++) {
            int vertn = csrRowOffset[j];
            if (vertn >= nvertices) continue;
            if (vertn != verti && aux[vertn] < INCLUDED) {
                aux[vertn] = aux[vertn] + NEIGHBOOR_COUNT_INCLUDED;
                if (aux[vertn] == INCLUDED) {
                    headQueue = MIN(headQueue, vertn);
                    tailQueue = MAX(tailQueue, vertn);
                }
            }
        }
        aux[verti] = PROCESSED;
        countIncluded++;
    }
    aux[vert] = SINCLUDED;
    return countIncluded;
}

__host__ __device__
int exapandHullSetFromV(int v, int nvertices, unsigned char *aux, unsigned char *auxb, int *graphData, int idx) {
    for (int j = 0; j < nvertices; j++) {
        aux[j] = 0;
    }
    if (verboseKernel) printf("thread-%d: cleanded aux\n", idx);
    if (verboseKernel) printf("thread-%d: add Vert %d to S\n", idx, v);

    int sizeHs = addVertToS(v, aux, graphData);
    int sSize = 1;
    int bv;
    do {
        bv = -1;
        int maiorGrau = 0;
        int maiorDeltaHs = 0;
        for (int i = 0; i < nvertices; i++) {
            if (aux[i] >= INCLUDED) {
                continue;
            }
            COPY_ARRAY(aux, auxb, nvertices);
            int deltaHsi = addVertToS(i, auxb, graphData);

            int neighborCount = 0;
            for (int j = 0; j < nvertices; j++) {
                if (auxb[j] == INCLUDED) {
                    neighborCount++;
                }
            }

            if (bv == -1 || (deltaHsi >= maiorDeltaHs && neighborCount > maiorGrau)) {
                maiorDeltaHs = deltaHsi;
                maiorGrau = neighborCount;
                bv = i;
            }
        }
        if (verboseKernel) printf("thread-%d: add Vert %d to S\n", idx, bv);
        sizeHs = sizeHs + addVertToS(bv, aux, graphData);
        sSize++;
    } while (sizeHs < nvertices);
    return sSize;
}

__host__ __device__
int exapandHullSetFromV(int v, int nvertices, unsigned char *aux, unsigned char *auxb, int *graphData, int idx, int *resultLocal) {
    for (int j = 0; j < nvertices; j++) {
        aux[j] = 0;
    }
    if (verboseKernel) printf("thread-%d: cleanded aux\n", idx);
    if (verboseKernel) printf("thread-%d: add Vert %d to S\n", idx, v);

    int sizeHs = addVertToS(v, aux, graphData);
    int sSize = 1;
    int bv;
    do {
        bv = -1;
        int maiorGrau = 0;
        int maiorDeltaHs = 0;
        for (int i = 0; i < nvertices; i++) {
            if (aux[i] >= INCLUDED) {
                continue;
            }
            COPY_ARRAY(aux, auxb, nvertices);
            int deltaHsi = addVertToS(i, auxb, graphData);

            int neighborCount = 0;
            for (int j = 0; j < nvertices; j++) {
                if (auxb[j] == INCLUDED) {
                    neighborCount++;
                }
            }

            if (bv == -1 || (deltaHsi >= maiorDeltaHs && neighborCount > maiorGrau)) {
                maiorDeltaHs = deltaHsi;
                maiorGrau = neighborCount;
                bv = i;
            }
        }
        if (verboseKernel) printf("thread-%d: add Vert %d to S\n", idx, bv);
        sizeHs = sizeHs + addVertToS(bv, aux, graphData);
        sSize++;
    } while (sizeHs < nvertices && sizeHs < *resultLocal);
    return sSize;
}

__global__
void kernelAproxHullNumberGraphByBlock(int* graphsGpu, int* dataGraphs, int* results) {
    //void kernelAproxHullNumber(graphCsr *graphs, int* results) {
    int offset = blockIdx.x;
    int idx = threadIdx.x;
    if (verboseKernel) printf("thread-%d\n", idx);

    int start = graphsGpu[offset];
    if (verboseKernel) printf("thread-%d Graph-start: %d\n", idx, start);

    int *graphData = &dataGraphs[start];
    int nvertices = graphData[0];

    if (idx == 0) {
        results[offset] = nvertices;
    }
    __syncthreads();

    if (verboseKernel) printf("thread-%d Graph-nvertices: %d\n", idx, nvertices);
    if (verboseKernel) printf("thread-%d Graph-edges: %d\n", idx, graphData[1]);


    if (idx < nvertices) {
        unsigned char *aux = new unsigned char [nvertices * 2];
        unsigned char *auxb = &aux[nvertices];

        int v = idx;
        int minHullSet = nvertices;
        int sSize = exapandHullSetFromV(v, nvertices, aux, auxb, graphData, idx);
        minHullSet = MIN(minHullSet, sSize);
        atomicMin(&results[offset], minHullSet);

        if (verboseKernel) printf("thread-%d: minHullSet %d\n", idx, minHullSet);

        free(aux);
        //    free(auxb);
        if (verboseKernel) printf("thread-%d: memory freed\n", idx);
    }
}

__global__
void kernelAproxHullNumberByVertex(int offset, int* graphsGpu,
        int* dataGraphs, int* results) {
    //void kernelAproxHullNumber(graphCsr *graphs, int* results) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (verboseKernel)
        printf("thread-%d\n", idx);

    int start = graphsGpu[offset];
    if (verboseKernel)
        printf("thread-%d Graph-start: %d\n", idx, start);

    int *graphData = &dataGraphs[start];
    int nvertices = graphData[0];

    if (idx == 0) {
        results[offset] = nvertices;
    }
    __syncthreads();

    if (idx < nvertices) {
        if (verboseKernel)
            printf("thread-%d Graph-nvertices: %d\n", idx, nvertices);
        if (verboseKernel)
            printf("thread-%d Graph-edges: %d\n", idx, graphData[1]);

        unsigned char *aux = new unsigned char [nvertices * 2];
        unsigned char *auxb = &aux[nvertices];
        int minHullSet = nvertices;
        int v = idx;

        int sSize = exapandHullSetFromV(v, nvertices, aux, auxb, graphData, idx);
        minHullSet = MIN(minHullSet, sSize);

        if (verboseKernel)
            printf("thread-%d: minHullSet %d\n", idx, minHullSet);
        if (verboseKernel)
            printf("thread-%d: memory freed\n", idx);
        atomicMin(&results[offset], minHullSet);
        free(aux);
    }
}

__global__
void kernelAproxHullNumberGraphByThread(int* graphsGpu, int* dataGraphs, int* results) {
    //void kernelAproxHullNumber(graphCsr *graphs, int* results) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (verboseKernel)
        printf("thread-%d\n", idx);

    int start = graphsGpu[idx];
    if (verboseKernel)
        printf("thread-%d Graph-start: %d\n", idx, start);

    int *graphData = &dataGraphs[start];
    int nvertices = graphData[0];

    if (verboseKernel)
        printf("thread-%d Graph-nvertices: %d\n", idx, nvertices);
    if (verboseKernel)
        printf("thread-%d Graph-edges: %d\n", idx, graphData[1]);

    unsigned char *aux = new unsigned char [nvertices * 2];
    //    unsigned char *auxb = new unsigned char [nvertices];
    unsigned char *auxb = &aux[nvertices];

    int minHullSet = nvertices;

    for (int v = 0; v < nvertices; v++) {
        for (int j = 0; j < nvertices; j++) {
            aux[j] = 0;
        }
        if (verboseKernel) printf("thread-%d: cleanded aux\n", idx);
        if (verboseKernel) printf("thread-%d: add Vert %d to S\n", idx, v);

        int sizeHs = addVertToS(v, aux, graphData);
        int sSize = 1;
        int bv;
        do {
            bv = -1;
            int maiorGrau = 0;
            int maiorDeltaHs = 0;
            for (int i = 0; i < nvertices; i++) {
                if (aux[i] >= INCLUDED) {
                    continue;
                }
                COPY_ARRAY(aux, auxb, nvertices);
                int deltaHsi = addVertToS(i, auxb, graphData);

                int neighborCount = 0;
                for (int j = 0; j < nvertices; j++) {
                    if (auxb[j] == INCLUDED) {
                        neighborCount++;
                    }
                }

                if (bv == -1 || (deltaHsi >= maiorDeltaHs && neighborCount > maiorGrau)) {
                    maiorDeltaHs = deltaHsi;
                    maiorGrau = neighborCount;
                    bv = i;
                }
            }
            if (verboseKernel) printf("thread-%d: add Vert %d to S\n", idx, bv);
            sizeHs = sizeHs + addVertToS(bv, aux, graphData);
            sSize++;
        } while (sizeHs < nvertices && sSize < minHullSet);
        minHullSet = MIN(minHullSet, sSize);
        if (verboseKernel) printf("thread-%d: minHullSet %d\n", idx, minHullSet);
    }
    free(aux);
    //    free(auxb);
    if (verboseKernel) printf("thread-%d: memory freed\n", idx);
    results[idx] = minHullSet;
}


//kernelAproxHullNumberGraphByBlockOptimal<<<numblocks,nthreads,maxgraphsbyblock*sizeof(int)>>>(graphsGpu, dataGraphsGpu, resultGpu, minvertice, maxvertice, totalvertices, maxgraphsbyblock);

__global__
void kernelAproxHullNumberGraphByBlockOptimal(int* graphsGpu, int* dataGraphs, int* results) {
    int igraph = blockIdx.x;
    int idx = threadIdx.x;

    __shared__ int resultLocal;

    if (idx == 0) {
        resultLocal = INT_MAX;
    }
    __syncthreads();

    if (verboseKernel) printf("thread-%d in block %d \n", idx, igraph);

    if (verboseKernel) printf("thread-%d Graph-start: %d\n", idx, igraph);

    int *graph = &dataGraphs[graphsGpu[igraph]];
    int nvertices = graph[0];



    if (verboseKernel) printf("thread-%d Graph-nvertices: %d\n", idx, nvertices);
    if (verboseKernel) printf("thread-%d Graph-edges: %d\n", idx, graph[1]);


    unsigned char *aux = new unsigned char [nvertices * 2];
    unsigned char *auxb = &aux[nvertices];

    int i = 0;
    while (idx < nvertices) {
        int v = idx;
        int minHullSet = nvertices;
        int sSize = exapandHullSetFromV(v, nvertices, aux, auxb, graph, idx, &resultLocal);
        //        int sSize = exapandHullSetFromV(v, nvertices, aux, auxb, graph, idx);

        minHullSet = MIN(minHullSet, sSize);
        //        atomicMin(&results[igraph], minHullSet);
        atomicMin(&resultLocal, minHullSet);

        if (verboseKernel) printf("thread-%d: minHullSet %d\n", idx, minHullSet);
        //    free(auxb);

        idx = idx + ((++i) * BLOCK_WINDOWS);
    }

    if (verboseKernel) printf("thread-%d: memory freed\n", idx);
    free(aux);

    __syncthreads();

    if (idx == 0) {
        results[igraph] = resultLocal;
        //        atomicMin(&results[igraph], resultLocal);
    }
}

//         << <numblocks, BLOCK_WINDOWS>>>(graphsGpu, dataGraphsGpu, resultGpu, nvertstrunk, mapworkgpu);

__global__
void kernelAproxHullNumberGraphByBlockOptimalTrunk(int* graphsGpu, int* dataGraphs, int* results, int *mapworkGpu, int nvertstrunk) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;


    //    if (idx == 0) {
    //        printf("\nigraph-initial=%d, offset=%d, maxn-vertices=%d", igraph, offset, nvertices);
    //        printf("\nmapk-work-gpu={");
    //        for (int a = 0; a < 2 * nvertstrunk; a++) {
    //            printf("%d, ", mapworkGpu[a]);
    //        }
    //        printf("}\n");
    //
    //        printf("\ngraphs-gpu={");
    //        for (int a = 0; a <= 3; a++) {
    //            printf("%d, ", graphsGpu[a]);
    //        }
    //        printf("}\n");
    //    }
    //
    //    __syncthreads();

    if (idx < nvertstrunk) {
        int igraph = mapworkGpu[0];
        int *graph = &dataGraphs[graphsGpu[igraph]];
        int nvertices = graph[0];


        int i = 0;
        while (idx < nvertstrunk) {
            int id = idx * 2;
            igraph = mapworkGpu[id];
            int v = mapworkGpu[id + 1];
            if (verboseKernel) printf("thread-%d in graph %d and vertice %d/%d \n", idx, igraph, v, nvertices);
            graph = &dataGraphs[graphsGpu[igraph]];
            nvertices = graph[0];

            unsigned char *aux = new unsigned char [nvertices * 2];
            unsigned char *auxb = &aux[nvertices];
            assert(aux != NULL);

            int minHullSet = nvertices;
            int sSize = exapandHullSetFromV(v, nvertices, aux, auxb, graph, idx);
            //            int sSize = exapandHullSetFromV(v, nvertices, aux, auxb, graph, idx, &results[igraph]);
            free(aux);

            minHullSet = MIN(minHullSet, sSize);
            atomicMin(&results[igraph], minHullSet);
            if (verboseKernel) printf("thread-%d: minHullSet %d\n", idx, minHullSet);
            idx = idx + ((++i) * BLOCK_WINDOWS_TRUNK);
        }
    }
}

int parallelAproxHullNumberGraphs(graphCsr *graphs, int cont) {
    if (verbose) printf("ParallelAproxHullNumberGraphs\n");
    int numbytesDataGraph = 0;
    hipError_t r;
    for (int i = 0; i < cont; i++) {
        int size = (graphs[i].data[0] + 3 + graphs[i].data[1]) * sizeof (int);
        numbytesDataGraph = numbytesDataGraph + size;
    }

    if (verbose) printf("Cuda Malloc Graph\n");

    int device;
    hipDeviceProp_t deviceProp;
    hipGetDevice(&device);
    hipGetDeviceProperties(&deviceProp, device);

    if ((deviceProp.concurrentKernels == 0)) {
        perror("not support for concurrent kernel\n");
    }
    if (verbose)
        printf("Compute SM %d.%d with %d multi-processors\n", deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount);

    int* dataGraphsGpu;
    int *graphsGpu;
    int *mapworkgpu;
    int* resultGpu;
    int* graphsHost = new int[cont];
    hipMalloc((void**) &resultGpu, cont * sizeof (int));
    hipMalloc((void**) &dataGraphsGpu, numbytesDataGraph);
    hipMalloc((void**) &graphsGpu, cont * sizeof (int));
    int offset = 0;
    int totalVert = 0;
    int currentCont = 0;

    //if (verbose) printf("Cuda Atrib Graph\n");

    //    if (verbose) {
    //printf("graph-gpu-offset={");
    //        printf("graph-gpu-data={");
    //    }
    for (int i = 0; i < cont; i++) {
        //if (verbose) printf("Cuda Atrib Graph-%d\n", i);
        graphCsr *graph = &graphs[i];
        int sizeGraph = graph->data[0] + 3 + graph->data[1];
        int nbytes = sizeGraph * sizeof (int);
        r = hipMemcpy(dataGraphsGpu + offset, graph->data, nbytes, hipMemcpyHostToDevice);
        if (r != hipSuccess) {
            fprintf(stderr, "Failed to copy memory 1 \nError: %s\n", hipGetErrorString(r));
            exit(EXIT_FAILURE);
        }

        //        if (verbose) printf("%d, ", offset);


        //        if (verbose) {
        //            for (int j = 0; j < sizeGraph; j++) {
        //                printf("%d, ", graph->data[j]);
        //            }
        //        }


        graphsHost[i] = offset;
        offset = offset + sizeGraph;

        currentCont = currentCont + (totalVert / BLOCK_WINDOWS);
        totalVert = (totalVert) % BLOCK_WINDOWS + graph->data[0];
    }
    //    if (verbose) {
    //        printf("}\n");
    //    }

    r = hipMemcpy(graphsGpu, graphsHost, cont * sizeof (int),
            hipMemcpyHostToDevice);
    if (r != hipSuccess) {
        fprintf(stderr, "Failed to copy memory 2 \nError: %s\n",
                hipGetErrorString(r));
        exit(EXIT_FAILURE);
    }


    hipStream_t *streams = (hipStream_t *) malloc(cont * sizeof (hipStream_t));
    for (int i = 0; i < cont; i++) {
        hipStreamCreate(&(streams[i]));
    }

    if (verbose) printf("Launch Kernel\n");

    int* resultLocal = new int[cont];

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    /* Graphs by Thread */
    if (graphByThread) {
        r = hipMemset(resultGpu, INT_MAX, sizeof (int)*cont);
        if (r != hipSuccess) {
            fprintf(stderr, "Failed to copy memory 4 \nError: %s\n", hipGetErrorString(r));
            exit(EXIT_FAILURE);
        }

        if (verbose) printf("Lanch Kernel One graph by thread\n");

        hipEventRecord(start);

        kernelAproxHullNumberGraphByThread << <1, cont>>>(graphsGpu, dataGraphsGpu, resultGpu);
        r = hipDeviceSynchronize();

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("Time for the kernel(one graph by thread): %2.f ms\n", milliseconds);

        if (verbose) printf("Read Result\n");

        if (r != hipSuccess) {
            fprintf(stderr, "Failed hipDeviceSynchronize \nError: %s\n", hipGetErrorString(r));
            exit(EXIT_FAILURE);
        }

        r = hipMemcpy(resultLocal, resultGpu, sizeof (int)*cont, hipMemcpyDeviceToHost);
        if (r != hipSuccess) {
            fprintf(stderr, "Failed to copy memory 4 \nError: %s\n", hipGetErrorString(r));
            exit(EXIT_FAILURE);
        }

        if (printResults)
            for (int i = 0; i < cont; i++) {
                printf("MinHullNumberAprox-Thread Graph-%d: %d\n", i, resultLocal[i]);
            }
    }

    /* Graphs by Block */
    if (graphByBlock) {
        r = hipMemset(resultGpu, INT_MAX, sizeof (int)*cont);
        if (r != hipSuccess) {
            fprintf(stderr, "Failed to copy memory 4 \nError: %s\n", hipGetErrorString(r));
            exit(EXIT_FAILURE);
        }

        if (verbose)
            printf("Lauch Kernel One graph by Block\n");
        hipEventRecord(start);

        int maxvertice = 0;
        for (int i = 0; i < cont; i++) {
            graphCsr *graph = &graphs[i];
            int nvertice = graph->data[0];
            if (nvertice > maxvertice) {
                maxvertice = nvertice;
            }
        }

        kernelAproxHullNumberGraphByBlock << <cont, maxvertice>>>(graphsGpu, dataGraphsGpu, resultGpu);

        //        r = hipDeviceSynchronize();

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("Time for the (one graph by block): %2.f ms\n", milliseconds);

        if (verbose)
            printf("Read Result\n");


        if (r != hipSuccess) {
            fprintf(stderr, "Failed hipDeviceSynchronize \nError: %s\n", hipGetErrorString(r));
            exit(EXIT_FAILURE);
        }

        r = hipMemcpy(resultLocal, resultGpu, sizeof (int)*cont, hipMemcpyDeviceToHost);
        if (r != hipSuccess) {
            fprintf(stderr, "Failed to copy memory 4 \nError: %s\n", hipGetErrorString(r));
            exit(EXIT_FAILURE);
        }

        if (printResults)
            for (int i = 0; i < cont; i++) {
                printf("MinHullNumberAprox-Block Graph-%d: %d\n", i, resultLocal[i]);
            }
    }

    /* Graph by Grid and Vertices by Thread */
    if (verticesBythread) {
        r = hipMemset(resultGpu, INT_MAX, sizeof (int)*cont);
        if (r != hipSuccess) {
            fprintf(stderr, "Failed to copy memory 4 \nError: %s\n", hipGetErrorString(r));
            exit(EXIT_FAILURE);
        }


        if (verbose)
            printf("Vertices by Thread One graph by kernel\n");
        hipEventRecord(start);

        for (int i = 0; i < cont; i++) {
            graphCsr *graph = &graphs[i];
            int nvertice = graph->data[0];
            int nblocks = nvertice / DEFAULT_THREAD_PER_BLOCK;
            if ((nvertice % DEFAULT_THREAD_PER_BLOCK) > 0) {
                nblocks++;
            }
            //            int nthreads = MIN(DEFAULT_THREAD_PER_BLOCK, nvertice);
            int nthreads = DEFAULT_THREAD_PER_BLOCK;
            kernelAproxHullNumberByVertex << <nblocks, nthreads, 0, streams[i]>>>(i, graphsGpu, dataGraphsGpu, resultGpu);
        }
        //        r = hipDeviceSynchronize();

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("Time for Vt/Thread kernel(one graph by kernel): %2.f ms\n", milliseconds);

        if (verbose)
            printf("Read Result\n");


        if (r != hipSuccess) {
            fprintf(stderr, "Failed hipDeviceSynchronize \nError: %s\n", hipGetErrorString(r));
            exit(EXIT_FAILURE);
        }


        r = hipMemcpy(resultLocal, resultGpu, sizeof (int)*cont, hipMemcpyDeviceToHost);
        if (r != hipSuccess) {
            fprintf(stderr, "Failed to copy memory 4 \nError: %s\n", hipGetErrorString(r));
            exit(EXIT_FAILURE);
        }

        if (printResults)
            for (int i = 0; i < cont; i++) {
                printf("MinHullNumberAprox-Vertices Thread-%d: %d\n", i, resultLocal[i]);
            }
    }

    /* Graphs by Block Optimal */
    if (graphByBlockOptimal) {
        r = hipMemset(resultGpu, INT_MAX, sizeof (int)*cont);
        if (r != hipSuccess) {
            fprintf(stderr, "Failed to copy memory 4 \nError: %s\n", hipGetErrorString(r));
            exit(EXIT_FAILURE);
        }


        if (verbose)
            printf("Lauch Kernel graph by Block optimal\n");

        int maxvertice = 0;
        int minvertice = INT_MAX;
        int totalvertices = 0;

        int nblocksoptimal = 0;
        int nblockstrunk = 0;
        int nvertstrunk = 0;

        for (int i = 0; i < cont; i++) {
            graphCsr *graph = &graphs[i];
            int nvertice = graph->data[0];

            if (nvertice >= BLOCK_WINDOWS) {
                nblocksoptimal++;
            }

            int vertrunk = nvertice % BLOCK_WINDOWS;
            nvertstrunk = nvertstrunk + vertrunk;
            if (vertrunk > 0) {
                nblockstrunk++;
            }

            if (nvertice > maxvertice) {
                maxvertice = nvertice;
            }
            if (nvertice < minvertice) {
                minvertice = nvertice;
            }
            totalvertices = totalvertices + nvertice;
        }

        int *mapwork = new int[2 * nvertstrunk];
        int j = 0;
        if (verbose) printf("mapwork={");

        for (int i = 0; i < cont; i++) {
            graphCsr *graph = &graphs[i];
            int nvertice = graph->data[0];

            int vertrunk = nvertice % BLOCK_WINDOWS;
            if (vertrunk > 0) {
                nblockstrunk++;
                for (int x = 0; x < vertrunk; x++) {
                    mapwork[j] = i;
                    int v = (nvertice / BLOCK_WINDOWS) * BLOCK_WINDOWS + x;
                    mapwork[j + 1] = v;
                    //                    printf("%d, %d, ", i, v);
                    j = j + 2;
                }
            }
        }

        if (verbose) printf("}\n");

        hipMalloc((void**) &mapworkgpu, 2 * nvertstrunk * sizeof (int));
        r = hipMemcpy(mapworkgpu, mapwork, 2 * nvertstrunk * sizeof (int), hipMemcpyHostToDevice);
        if (r != hipSuccess) {
            fprintf(stderr, "Failed to copy memory 1.2 \nError: %s\n", hipGetErrorString(r));
            exit(EXIT_FAILURE);
        }

        //https://devtalk.nvidia.com/default/topic/1026825/how-to-choose-how-many-threads-blocks-to-have-/
        //Therefore very small block sizes (e.g. 32 threads per block) may limit performance due to occupancy. 
        //Very large block sizes for example 1024 threads per block, may also limit performance, if there are resource limits 
        //(e.g. registers per thread usage, or shared memory usage) which prevent 2 threadblocks (in this example of 1024 threads per block) from being resident on a SM
        //Threadblock size choices in the range of 128 - 512 are less likely to run into the aforementioned issues


        int numblocks = nvertstrunk / BLOCK_WINDOWS_TRUNK;
        if ((nvertstrunk % BLOCK_WINDOWS_TRUNK) > 0) {
            numblocks++;
        }

        printf("Starting kernel optimal: <%d,%d>\n", nblocksoptimal, BLOCK_WINDOWS);
        printf("Starting kernel turnk: <%d,%d>\n", numblocks, BLOCK_WINDOWS_TRUNK);


        hipEventRecord(start);

        kernelAproxHullNumberGraphByBlockOptimal << <nblocksoptimal, BLOCK_WINDOWS, nblocksoptimal * sizeof (int), streams[0]>>>(graphsGpu, dataGraphsGpu, resultGpu);
        kernelAproxHullNumberGraphByBlockOptimalTrunk << <numblocks, BLOCK_WINDOWS_TRUNK, 0, streams[1]>>>(graphsGpu, dataGraphsGpu, resultGpu, mapworkgpu, nvertstrunk);
        //        kernelAproxHullNumberGraphByBlockOptimalTrunk << <numblocks, nthreads>>>(mapworkgpu, graphsGpu, cont, dataGraphsGpu, resultGpu, minvertice, maxvertice, totalvertices, maxgraphsbyblock);
        //        kernelAproxHullNumberGraphByBlockOptimal << <numblocks, nthreads, maxgraphsbyblock * sizeof (int)>>>(mapworkgpu, graphsGpu, cont, dataGraphsGpu, resultGpu, minvertice, maxvertice, totalvertices, maxgraphsbyblock);
        //        r = hipDeviceSynchronize();

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("Time for the kernel(graph by block optimal): %2.f ms\n", milliseconds);

        if (verbose)
            printf("Read Result\n");


        if (r != hipSuccess) {
            fprintf(stderr, "Failed hipDeviceSynchronize \nError: %s\n", hipGetErrorString(r));
            exit(EXIT_FAILURE);
        }


        r = hipMemcpy(resultLocal, resultGpu, sizeof (int)*cont, hipMemcpyDeviceToHost);
        if (r != hipSuccess) {
            fprintf(stderr, "Failed to copy memory 4 \nError: %s\n", hipGetErrorString(r));
            exit(EXIT_FAILURE);
        }

        if (printResults)
            for (int i = 0; i < cont; i++) {
                printf("MinHullNumberAprox-Block Graph  optimal-%d: %d\n", i, resultLocal[i]);
            }

        free(mapwork);
    }

    r = hipGetLastError();
    if (r != hipSuccess) {
        fprintf(stderr, "Failed in kernelAproxHullNumber \nError: %s\n", hipGetErrorString(r));
        exit(EXIT_FAILURE);
    }


    free(streams);
    free(resultLocal);
    free(graphsHost);
    hipFree(mapworkgpu);
    hipFree(resultGpu);
    hipFree(graphsGpu);
    hipFree(dataGraphsGpu);
}

int serialAproxHullNumber(int *graphData) {
    int nvertices = graphData[0];
    //    printf("Nvertices: %d\n", nvertices);
    unsigned char *aux = (unsigned char *) malloc(sizeof (unsigned char)*nvertices);
    unsigned char *auxb = (unsigned char *) malloc(sizeof (unsigned char)*nvertices);
    if (aux == NULL || auxb == NULL) perror("Not memory allocation");
    int minHullSet = nvertices;

    for (int v = 0; v < nvertices; v++) {
        for (int j = 0; j < nvertices; j++) {
            aux[j] = 0;
        }

        int sizeHs = addVertToS(v, aux, graphData);
        int sSize = 1;
        int bv;
        do {
            bv = -1;
            int maiorGrau = 0;
            int maiorDeltaHs = 0;
            for (int i = 0; i < nvertices; i++) {
                if (aux[i] >= INCLUDED) {
                    continue;
                }
                COPY_ARRAY(aux, auxb, nvertices);
                int deltaHsi = addVertToS(i, auxb, graphData);

                int neighborCount = 0;
                for (int j = 0; j < nvertices; j++) {
                    if (auxb[j] == INCLUDED) {
                        neighborCount++;
                    }
                }

                if (bv == -1 || (deltaHsi >= maiorDeltaHs && neighborCount > maiorGrau)) {
                    maiorDeltaHs = deltaHsi;
                    maiorGrau = neighborCount;
                    bv = i;
                }
            }
            sizeHs = sizeHs + addVertToS(bv, aux, graphData);
            sSize++;
        } while (sizeHs < nvertices && sSize < minHullSet);
        minHullSet = MIN(minHullSet, sSize);
    }
    free(aux);
    free(auxb);
    return minHullSet;
}

void serialAproxHullNumberGraphs(graphCsr *graphs, int cont) {
    if (verbose) printf("Serial execution \n");
    int *results = new int[cont];
    clock_t begin_serial_time, end_serial_time;
    begin_serial_time = clock();
    for (int i = 0; i < cont; i++) {
        results[i] = serialAproxHullNumber(graphs[i].data);
        //        printf("MinHullNumberAprox-Serial Graph-%d: %d\n", i, results[i]);
    }
    end_serial_time = clock();

    double diff = end_serial_time - begin_serial_time;
    diff = diff / (CLOCKS_PER_SEC / 1000);

    printf("Time for the serial: %2.f ms\n", diff);


    if (printResults)
        for (int i = 0; i < cont; i++) {
            printf("MinHullNumberAprox-Serial Graph-%d: %d\n", i, results[i]);
        }

    free(results);
}

void processFiles(int argc, char** argv) {
    graphCsr* graphs = (graphCsr*) malloc((sizeof (graphCsr)) * argc);
    std::vector<int> values;

    int contGraph = 0;
    int verticesMedian = 0;

    for (int x = 1; x < argc; x++) {
        char* strFile = argv[x];
        DIR *dpdf;
        struct dirent *epdf;
        struct stat filestat;

        if (strFile[0] == '-') {
            continue;
        }

        dpdf = opendir(strFile);
        std::string filepath = std::string(strFile);

        while (dpdf && (epdf = readdir(dpdf))) {
            filepath = std::string(strFile) + "/" + epdf->d_name;
            if (epdf->d_name == "." || epdf->d_name == "..")
                continue;
            if (stat(filepath.c_str(), &filestat))
                continue;
            if (S_ISDIR(filestat.st_mode))
                continue;
            else break;
        }
        closedir(dpdf);

        std::string line, strCArray, strRArray;
        std::ifstream infile(filepath.c_str());

        if (infile) {
            while (getline(infile, line)) {
                if (line.at(0) != CHARACTER_INIT_COMMENT && line.at(1) != CHARACTER_INIT_COMMENT) {
                    if (strCArray.empty()) {
                        strCArray = line;
                    } else if (strRArray.empty()) {
                        strRArray = line;
                    }
                }
            }
            infile.close();
        } else {
            continue;
        }

        if (strCArray.empty() || strRArray.empty()) {
            perror("Invalid file format");
            continue;
        }

        std::stringstream stream(strCArray.c_str());
        values.clear();

        int n;
        while (stream >> n) {
            values.push_back(n);
        }
        strCArray.clear();

        int numVertices = values.size() - 1;
        stream.str("");

        std::stringstream stream2(strRArray);
        while (stream2 >> n) {
            values.push_back(n);
        }
        stream2.str("");
        strRArray.clear();

        int numedges = values.size() - (numVertices + 1);
        values.insert(values.begin(), numedges);
        values.insert(values.begin(), numVertices);
        int *data = new int[values.size()];
        std::copy(values.begin(), values.end(), data);

        values.clear();

        if (numVertices < 1) {
            continue;
        }

        graphCsr* graph = &graphs[contGraph];
        graph->data = data;
        contGraph++;
        verticesMedian = verticesMedian + numVertices;
    }


    qsort(graphs, contGraph, sizeof (struct graphCsr), compare);

    if (contGraph > 0)
        printf("Processing: %d graphs  %dv/g (avg) \n", contGraph, verticesMedian / contGraph);

    //    if (verbose) {
    //        for (int i = 0; i < contGraph; i++) {
    //            printf("%d, ", graphs[i].data[0]);
    //        }
    //    }

    if (graphByBlock || graphByThread || verticesBythread || graphByBlockOptimal)
        parallelAproxHullNumberGraphs(graphs, contGraph);

    if (serial)
        serialAproxHullNumberGraphs(graphs, contGraph);

    for (int i = 0; i < contGraph; i++) {
        free(graphs[i].data);
    }
    free(graphs);
}

void runTest() {
    //    int numVertices = 10;
    //    int colIdx[] = {0, 3, 6, 9, 12, 15, 18, 21, 24, 27, 30};
    //    int sizeRowOffset = numVertices + 1;
    //    int rowOffset[] = {2, 5, 6, 3, 4, 6, 0, 4, 7, 1, 5, 7, 1, 2, 9, 0, 3, 9, 0, 1, 8, 2, 3, 8, 6, 7, 9, 4, 5, 8};

    printf("graphHullNumberAprox -[svtbnx] dir\n");

    int data[] = {10, 30,
        0, 3, 6, 9, 12, 15, 18, 21, 24, 27, 30,
        2, 5, 6, 3, 4, 6, 0, 4, 7, 1, 5, 7, 1, 2, 9, 0, 3, 9, 0, 1, 8, 2, 3, 8, 6, 7, 9, 4, 5, 8};
    graphCsr* graph = (graphCsr*) malloc(sizeof (graphCsr));
    graph->data = data;
    parallelAproxHullNumberGraphs(graph, 1);
}

int main(int argc, char** argv) {
    printf("Main\n");

    long opt = 0;

    while ((opt = getopt(argc, argv, "svtbnxo")) != -1) {
        switch (opt) {
            case 't':
                graphByThread = true;
                break;
            case 'b':
                graphByBlock = true;
                break;
            case 's':
                serial = true;
                break;
            case 'v':
                verbose = true;
                break;
            case 'x':
                verticesBythread = true;
                break;
            case 'o':
                graphByBlockOptimal = true;
                break;
            case 'n':
                printResults = false;
                break;
            case '?':
                printf("Unknow option: %c", char(opt));
                break;
        }
    }

    if (argc > 1) {
        processFiles(argc, argv);
    } else {
        runTest();
    }
    return 0;
}